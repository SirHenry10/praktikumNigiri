#pragma once

#include "nigiri/routing/gpu_raptor_state.h"

#include <hip/hip_runtime.h>

std::pair<dim3, dim3> get_launch_paramters(
    hipDeviceProp_t const& prop, int32_t const concurrency_per_device) {
  int32_t block_dim_x = 32;  // must always be 32!
  int32_t block_dim_y = 32;  // range [1, ..., 32]
  int32_t block_size = block_dim_x * block_dim_y;
  int32_t max_blocks_per_sm = prop.maxThreadsPerMultiProcessor / block_size;

  auto const mp_count = prop.multiProcessorCount / concurrency_per_device;

  int32_t num_blocks = mp_count * max_blocks_per_sm;

  dim3 threads_per_block(block_dim_x, block_dim_y, 1);
  dim3 grid(num_blocks, 1, 1);

  return {threads_per_block, grid};
}

device_context::device_context(device_id const device_id)
    : id_(device_id) {
  hipSetDevice(id_);
  cuda_check();

  hipGetDeviceProperties(&props_, device_id);
  cuda_check();

  std::tie(threads_per_block_, grid_) =
      get_launch_paramters(props_, 1);

  hipStreamCreate(&proc_stream_);
  cuda_check();
  hipStreamCreate(&transfer_stream_);
  cuda_check();
}

void device_context::destroy() {
  hipSetDevice(id_);
  hipStreamDestroy(proc_stream_);
  proc_stream_ = hipStream_t{};
  hipStreamDestroy(transfer_stream_);
  transfer_stream_ = hipStream_t{};
  cuda_check();
}

// Attribute, die von Host benötigt werden
host_memory::host_memory(uint32_t row_count_round_times,
                         uint32_t column_count_round_times,
                         uint32_t n_locations,
                         uint32_t n_routes
                         ):row_count_round_times_{row_count_round_times},
                             column_count_round_times_{column_count_round_times},
round_times_{std::vector<gpu_delta_t>(row_count_round_times*column_count_round_times)},
stats_{std::vector<gpu_raptor_stats>(n_locations)},tmp_{std::vector<gpu_delta_t>(n_locations)},best_{std::vector<gpu_delta_t>(n_locations)},station_mark_{std::vector<uint32_t>(n_locations)},
      prev_station_mark_{std::vector<uint32_t>(n_locations)},route_mark_{std::vector<uint32_t>(n_routes)}{}

// Zuweisung von Speicherplatz an Attribute, die in devices verwendet werden
device_memory::device_memory(uint32_t n_locations,
                             uint32_t n_routes,
                             uint32_t row_count_round_times,
                             uint32_t column_count_round_times,
                             gpu_delta_t invalid)
    : n_locations_{n_locations},
      n_routes_{n_routes},
      row_count_round_times_{row_count_round_times},
      column_count_round_times_{column_count_round_times}{
  hipMalloc(&tmp_, n_locations_ * sizeof(gpu_delta_t));
  time_at_dest_ = nullptr;
  hipMalloc(&time_at_dest_, (gpu_kMaxTransfers+1) *sizeof(gpu_delta_t));
  hipMalloc(&best_, n_locations_ * sizeof(gpu_delta_t));
  hipMalloc(&round_times_, row_count_round_times_ * column_count_round_times_ *
                                sizeof(gpu_delta_t));
  hipMalloc(&station_mark_, n_locations_ * sizeof(uint32_t));
  hipMalloc(&prev_station_mark_, n_locations_ * sizeof(uint32_t));
  hipMalloc(&route_mark_, n_routes_ * sizeof(uint32_t));
  hipMalloc(&any_station_marked_, sizeof(bool));
  hipMalloc(&stats_,32*sizeof(gpu_raptor_stats));
  invalid_ = invalid;
  cuda_check();
  this->reset_async(nullptr);
}

void device_memory::destroy() {
  hipFree(time_at_dest_);
  hipFree(tmp_);
  hipFree(best_);
  hipFree(round_times_);
  hipFree(station_mark_);
  hipFree(prev_station_mark_);
  hipFree(route_mark_);
  hipFree(stats_);
}

void device_memory::reset_async(hipStream_t s) {
  hipMemsetAsync(time_at_dest_,invalid_, (gpu_kMaxTransfers+1)*sizeof(gpu_delta_t), s);
  hipMemsetAsync(tmp_,invalid_, n_locations_*sizeof(gpu_delta_t), s);
  hipMemsetAsync(best_, invalid_, n_locations_*sizeof(gpu_delta_t), s);
  hipMemsetAsync(round_times_, invalid_, column_count_round_times_*row_count_round_times_*sizeof(gpu_delta_t), s);
  hipMemsetAsync(station_mark_, 0, n_locations_*sizeof(uint32_t), s);
  hipMemsetAsync(prev_station_mark_, 0, n_locations_*sizeof(uint32_t), s);
  hipMemsetAsync(route_mark_, 0, n_routes_*sizeof(uint32_t), s);
  hipMemsetAsync(any_station_marked_, 0, sizeof(bool), s);
  gpu_raptor_stats init_value = {};
  for (int i = 0; i < 32; ++i) {
    hipMemcpyAsync(&stats_[i], &init_value, sizeof(gpu_raptor_stats), hipMemcpyHostToDevice, s);
  }
  //additional_start_count_ = invalid<decltype(additional_start_count_)>;
}
void device_memory::next_start_time_async(hipStream_t s) {
  hipMemsetAsync(tmp_,invalid_, n_locations_*sizeof(gpu_delta_t), s);
  hipMemsetAsync(best_, invalid_, n_locations_*sizeof(gpu_delta_t), s);
  hipMemsetAsync(station_mark_, 0, n_locations_*sizeof(uint32_t), s);
  hipMemsetAsync(prev_station_mark_, 0, n_locations_*sizeof(uint32_t), s);
  hipMemsetAsync(route_mark_, 0, n_routes_*sizeof(uint32_t), s);
}
void device_memory::reset_arrivals_async(hipStream_t s) {
  hipMemsetAsync(time_at_dest_,invalid_, (gpu_kMaxTransfers+1)*sizeof(gpu_delta_t), s);
  hipMemsetAsync(round_times_, invalid_, column_count_round_times_*row_count_round_times_*sizeof(gpu_delta_t), s);
}
mem::mem(uint32_t n_locations, uint32_t n_routes,
         uint32_t row_count_round_times_, uint32_t column_count_round_times_,gpu_delta_t invalid,
         device_id const device_id)
    : host_{row_count_round_times_, column_count_round_times_,n_locations,n_routes},
      device_{n_locations, n_routes, row_count_round_times_, column_count_round_times_, invalid},
      context_{device_id} {}

mem::~mem() {
  device_.destroy();
  context_.destroy();
}

void gpu_raptor_state::init(gpu_timetable const& gtt,gpu_delta_t invalid) {
  int32_t device_count = 0;
  hipGetDeviceCount(&device_count);


  for (auto device_id = 0; device_id < device_count; ++device_id) {
      memory_.emplace_back(std::make_unique<struct mem>(
        *gtt.n_locations_,*gtt.n_routes_,gpu_kMaxTransfers + 1U,*gtt.n_locations_,invalid, device_id));
  }
  memory_mutexes_ = std::vector<std::mutex>(memory_.size());
}

gpu_raptor_state::mem_idx gpu_raptor_state::get_mem_idx() {
  return current_idx_.fetch_add(1) % memory_.size();
}


loaned_mem::loaned_mem(gpu_raptor_state& store,gpu_delta_t invalid) {
  auto const idx = store.get_mem_idx();
  lock_ = std::unique_lock(store.memory_mutexes_[idx]);
  mem_ = store.memory_[idx].get();
  mem_->device_.invalid_ = invalid;
}

loaned_mem::~loaned_mem() {
  mem_->device_.reset_async(mem_->context_.proc_stream_);
  cuda_sync_stream(mem_->context_.proc_stream_);
}
void mem::reset_arrivals_async(){
  device_.reset_arrivals_async(context_.proc_stream_);
  cuda_sync_stream(context_.proc_stream_);
}
void mem::next_start_time_async(){
  device_.next_start_time_async(context_.proc_stream_);
  cuda_sync_stream(context_.proc_stream_);
}