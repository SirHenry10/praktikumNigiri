#pragma once

#include <hip/hip_runtime.h>
#include "nigiri/routing/gpu_timetable.h"
#include <cstdio>

#define XSTR(s) STR(s)
#define STR(s) #s

#define CUDA_CALL(call)                                   \
    if ((code = call) != hipSuccess) {                     \
      printf("CUDA error: %s at " STR(call) " %s:%d\n",     \
             hipGetErrorString(code), __FILE__, __LINE__); \
      goto fail;                                            \
    }

#define CUDA_COPY_TO_DEVICE(type, target, source, size)                        \
    CUDA_CALL(hipMalloc(&target, size * sizeof(type)))                          \
    CUDA_CALL(                                                                   \
        hipMemcpy(target, source, size * sizeof(type), hipMemcpyHostToDevice)) \
    device_bytes += size * sizeof(type);
#define CUDA_CALL_VECVEC(call)                                   \
    if ((code = call) != hipSuccess) {                     \
      printf("CUDA error: %s at " STR(call) " %s:%d\n",     \
             hipGetErrorString(code), __FILE__, __LINE__); \
      goto fail_vecvec;                                            \
    }

template <typename KeyType, typename ValueType>
gpu_vecvec<KeyType, ValueType>* copy_gpu_vecvec_to_device(gpu_vecvec<KeyType, ValueType> const* host_vecvec, size_t& device_bytes, hipError_t& code) {
  gpu_vecvec<KeyType, ValueType>* device_vecvec = nullptr;
  ValueType* device_data = nullptr;
  KeyType* device_bucket_starts = nullptr;

  CUDA_CALL_VECVEC(hipMalloc(&device_vecvec, sizeof(gpu_vecvec<KeyType, ValueType>)));
  CUDA_CALL_VECVEC(hipMemcpy(device_vecvec, host_vecvec, sizeof(gpu_vecvec<KeyType, ValueType>), hipMemcpyHostToDevice));
  device_bytes += sizeof(gpu_vecvec<KeyType, ValueType>);

  CUDA_CALL_VECVEC(hipMalloc(&device_data, host_vecvec->data_.size() * sizeof(ValueType)));
  CUDA_CALL_VECVEC(hipMemcpy(device_data, host_vecvec->data_.data(), host_vecvec->data_.size() * sizeof(ValueType), hipMemcpyHostToDevice));
  device_bytes += host_vecvec->data_.size() * sizeof(ValueType);

  CUDA_CALL_VECVEC(hipMalloc(&device_bucket_starts, host_vecvec->bucket_starts_.size() * sizeof(KeyType)));
  CUDA_CALL_VECVEC(hipMemcpy(device_bucket_starts, host_vecvec->bucket_starts_.data(), host_vecvec->bucket_starts_.size() * sizeof(KeyType), hipMemcpyHostToDevice));
  device_bytes += host_vecvec->bucket_starts_.size() * sizeof(KeyType);

  CUDA_CALL_VECVEC(hipMemcpy(&(device_vecvec->data_), &device_data, sizeof(ValueType*), hipMemcpyHostToDevice));
  CUDA_CALL_VECVEC(hipMemcpy(&(device_vecvec->bucket_starts_), &device_bucket_starts, sizeof(KeyType*), hipMemcpyHostToDevice));

  return device_vecvec;

fail_vecvec:
  if (device_data) hipFree(device_data);
  if (device_bucket_starts) hipFree(device_bucket_starts);
  if (device_vecvec) hipFree(device_vecvec);
  return nullptr;
}

struct gpu_timetable* create_gpu_timetable(gpu_delta const* route_stop_times,
                                           std::uint32_t  n_route_stop_times,
                                           gpu_vecvec<gpu_route_idx_t,gpu_value_type> const* route_location_seq,
                                           gpu_vecvec<gpu_location_idx_t , gpu_route_idx_t> const* location_routes,
                                           std::uint32_t const* n_locations,
                                           std::uint32_t const* n_routes,
                                           gpu_vector_map<gpu_route_idx_t,gpu_interval<std::uint32_t>> const* route_stop_time_ranges,
                                           gpu_vector_map<gpu_route_idx_t,gpu_interval<gpu_transport_idx_t >> const* route_transport_ranges,
                                           gpu_vector_map<gpu_bitfield_idx_t, gpu_bitfield> const* bitfields,
                                           gpu_vector_map<gpu_bitfield_idx_t,std::uint64_t*> const* bitfields_data,
                                           gpu_vector_map<gpu_transport_idx_t,gpu_bitfield_idx_t> const* transport_traffic_days,
                                           gpu_interval<gpu_sys_days> const* date_range,
                                           gpu_locations const* locations,
                                           gpu_vector_map<gpu_route_idx_t, gpu_clasz> const* route_clasz) {
  size_t device_bytes = 0U;

  hipError_t code;
  gpu_timetable* gtt =
      static_cast<gpu_timetable*>(malloc(sizeof(gpu_timetable)));
  if (gtt == nullptr) {
    printf("nigiri gpu raptor: malloc for gpu_timetable failed\n");
    return nullptr;
  }
  // route_stop_times_
  gtt->route_stop_times_ = nullptr;
  CUDA_COPY_TO_DEVICE(gpu_delta, gtt->route_stop_times_, route_stop_times,
                      n_route_stop_times);
  //route_location_seq
  gtt->route_location_seq_ = copy_gpu_vecvec_to_device(route_location_seq,device_bytes,code);
  if (gtt->route_location_seq_ == nullptr) goto fail;
  //location_routes_
  gtt->location_routes_ = copy_gpu_vecvec_to_device(location_routes, device_bytes, code);
  if (gtt->location_routes_ == nullptr) goto fail;
  //n_locations_
  gtt->n_locations_ = nullptr;
  CUDA_COPY_TO_DEVICE(uint32_t , gtt->n_locations_, n_locations,1);
  //n_routes_
  gtt->n_routes_ = nullptr;
  CUDA_COPY_TO_DEVICE(uint32_t , gtt->n_routes_, n_routes,1);
  //route_stop_time_ranges_
  gtt->route_stop_time_ranges_ = nullptr;
  using gpu_vecmap_stop_time_ranges = gpu_vector_map<gpu_route_idx_t,gpu_interval<std::uint32_t>>;
  CUDA_COPY_TO_DEVICE(gpu_vecmap_stop_time_ranges , gtt->route_stop_time_ranges_, route_stop_time_ranges,1);
  //route_transport_ranges_
  gtt->route_transport_ranges_ = nullptr;
  using gpu_vecmap_route_transport_ranges = gpu_vector_map<gpu_route_idx_t,gpu_interval<gpu_transport_idx_t >>;
  CUDA_COPY_TO_DEVICE(gpu_vecmap_route_transport_ranges , gtt->route_transport_ranges_, route_transport_ranges,1);
  //bitfields_
  gtt->bitfields_ = nullptr;
  using gpu_vecmap_bitfields = gpu_vector_map<gpu_bitfield_idx_t, gpu_bitfield>;
  CUDA_COPY_TO_DEVICE(gpu_vecmap_bitfields, gtt->bitfields_, bitfields,1);
  //bitfields_data_
  gtt->bitfields_data_ = nullptr;
  using gpu_vecmap_bitfields_data = gpu_vector_map<gpu_bitfield_idx_t, std::uint64_t*>;
  CUDA_COPY_TO_DEVICE(gpu_vecmap_bitfields_data, gtt->bitfields_data_, bitfields_data,1);
  //transport_traffic_days_
  gtt->transport_traffic_days_ = nullptr;
  using gpu_vecmap_transport_traffic_days = gpu_vector_map<gpu_transport_idx_t,gpu_bitfield_idx_t>;
  CUDA_COPY_TO_DEVICE(gpu_vecmap_transport_traffic_days, gtt->transport_traffic_days_, transport_traffic_days,1);
  //date_range_
  gtt->date_range_ = nullptr;
  using gpu_date_range = gpu_interval<gpu_sys_days>;
  CUDA_COPY_TO_DEVICE(gpu_date_range , gtt->date_range_, date_range,1);
  //locations_
  gtt->locations_ = nullptr;
  CUDA_COPY_TO_DEVICE(gpu_locations , gtt->locations_, locations,1);
  //route_clasz_
  gtt->route_clasz_ = nullptr;
  using gpu_vector_map_clasz = gpu_vector_map<gpu_route_idx_t, gpu_clasz>;
  CUDA_COPY_TO_DEVICE(gpu_vector_map_clasz, gtt->route_clasz_, route_clasz,1);
  return gtt;


fail:
  hipFree(gtt->route_stop_times_);
  hipFree(gtt->route_location_seq_);
  hipFree(gtt->location_routes_);
  hipFree(gtt->n_locations_);
  hipFree(gtt->n_routes_);
  hipFree(gtt->route_stop_time_ranges_);
  hipFree(gtt->route_transport_ranges_);
  hipFree(gtt->bitfields_);
  hipFree(gtt->bitfields_data_);
  hipFree(gtt->transport_traffic_days_);
  hipFree(gtt->date_range_);
  hipFree(gtt->locations_);
  hipFree(gtt->route_clasz_);
  free(gtt);
  return nullptr;
}
void destroy_gpu_timetable(gpu_timetable* gtt) {
  hipFree(gtt->route_stop_times_);
  hipFree(gtt->route_location_seq_->data_.data());
  hipFree(gtt->route_location_seq_->bucket_starts_.data());
  hipFree(gtt->route_location_seq_);
  hipFree(gtt->location_routes_->data_.data());
  hipFree(gtt->location_routes_->bucket_starts_.data());
  hipFree(gtt->location_routes_);
  hipFree(gtt->n_locations_);
  hipFree(gtt->n_routes_);
  hipFree(gtt->route_stop_time_ranges_);
  hipFree(gtt->route_transport_ranges_);
  hipFree(gtt->bitfields_);
  hipFree(gtt->bitfields_data_);
  hipFree(gtt->transport_traffic_days_);
  hipFree(gtt->date_range_);
  hipFree(gtt->locations_);
  hipFree(gtt->route_clasz_);
  free(gtt);
  gtt = nullptr;
  hipDeviceSynchronize();
  auto const last_error = hipGetLastError();
  if (last_error != hipSuccess) {
    printf("CUDA error: %s at " STR(last_error) " %s:%d\n",
           hipGetErrorString(last_error), __FILE__, __LINE__);
  }
}