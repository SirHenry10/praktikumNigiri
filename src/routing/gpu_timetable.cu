#pragma once

#include <hip/hip_runtime.h>
#include "nigiri/routing/gpu_timetable.h"
#include <iostream>
#include <cstdio>

#define XSTR(s) STR(s)
#define STR(s) #s

#define CUDA_CALL(call)                                   \
    if ((code = call) != hipSuccess) {                     \
      printf("CUDA error: %s at " STR(call) " %s:%d\n",     \
             hipGetErrorString(code), __FILE__, __LINE__); \
      goto fail;                                            \
    }

#define CUDA_COPY_TO_DEVICE(type, target, source, size)                        \
    CUDA_CALL(hipMalloc(&target, size * sizeof(type)))                          \
    CUDA_CALL(                                                                   \
        hipMemcpy(target, source, size * sizeof(type), hipMemcpyHostToDevice)) \
    device_bytes += size * sizeof(type);

template <typename KeyType, typename ValueType>
gpu_vecvec<KeyType, ValueType>* copy_gpu_vecvec_to_device(gpu_vecvec<KeyType, ValueType> const* host_vecvec, size_t& device_bytes, hipError_t& code) {
  std::cerr << "Test3.11" << std::endl;
  fflush(stdout);
  using VecVec = gpu_vecvec<KeyType, ValueType>;

  // Step 1: Allocate memory on the device for the `gpu_vecvec` structure
  VecVec* device_vecvec = nullptr;
  gpu_base_t<KeyType>* device_bucket_starts = nullptr;
  ValueType* device_data = nullptr;
  VecVec host_vecvec_copy = *host_vecvec; // Create a host copy to modify the pointers
  CUDA_CALL(hipMalloc(&device_vecvec, sizeof(VecVec)));

  std::cerr << "Test3.112" << std::endl;
  // Step 2: Allocate memory on the device for `bucket_starts`
  CUDA_CALL(hipMalloc(&device_bucket_starts, host_vecvec->bucket_starts_.size() * sizeof(gpu_base_t<KeyType>)));

  std::cerr << "Test3.113" << std::endl;
  // Copy `bucket_starts` from host to device
  CUDA_CALL(hipMemcpy(device_bucket_starts, host_vecvec->bucket_starts_.data(),
                    host_vecvec->bucket_starts_.size() * sizeof(gpu_base_t<KeyType>), hipMemcpyHostToDevice));

  std::cerr << "Test3.114" << std::endl;
  // Step 3: Allocate memory on the device for `data`
  CUDA_CALL(hipMalloc(&device_data, host_vecvec->data_.size() * sizeof(ValueType)));

  std::cerr << "Test3.115" << std::endl;
  // Copy `data` from host to device
  CUDA_CALL(hipMemcpy(device_data, host_vecvec->data_.data(),
                    host_vecvec->data_.size() * sizeof(ValueType), hipMemcpyHostToDevice));

  std::cerr << "Test3.116" << std::endl;
  // Step 4: Update the `gpu_vecvec` on the device to point to device memory
  host_vecvec_copy.bucket_starts_.el_ = device_bucket_starts;
  host_vecvec_copy.data_.el_ = device_data;

  std::cerr << "Test3.117" << std::endl;
  // Copy the modified `gpu_vecvec` from host to device
  CUDA_CALL(hipMemcpy(device_vecvec, &host_vecvec_copy, sizeof(VecVec), hipMemcpyHostToDevice));

  std::cerr << "Test3.118" << std::endl;
  device_bytes += sizeof(VecVec);
  device_bytes += host_vecvec->bucket_starts_.size() * sizeof(gpu_base_t<KeyType>);
  device_bytes += host_vecvec->data_.size() * sizeof(ValueType);

  if (hipDeviceSynchronize() != hipSuccess) {
    std::cerr << "CUDA synchronization error "  << std::endl;
    goto fail;
  }
  std::cerr << "Test3.119" << std::endl;
  if (!device_vecvec || !device_bucket_starts || !device_data) {
    std::cerr << "Error: Memory allocation failed." << std::endl;
    goto fail;
  }
  std::cerr << "device_data_data: " << host_vecvec_copy.data_.data() << std::endl;
  std::cerr << "device_bucket_starts_data: " << host_vecvec_copy.bucket_starts_.data() << std::endl;
  std::cerr << "device_data_data: " << device_vecvec->data_.data() << std::endl;
  std::cerr << "device_bucket_starts_data: " << device_vecvec->bucket_starts_.data() << std::endl;
  std::cerr << "Pointer status: " << std::endl;
  std::cerr << "device_vecvec: " << device_vecvec << std::endl;
  std::cerr << "device_bucket_starts: " << device_bucket_starts << std::endl;
  std::cerr << "device_data: " << device_data << std::endl;
  std::cerr << "device_data_size: " << device_vecvec->data_.size() << std::endl;
  std::cerr << "device_bucket_starts_size: " << device_vecvec->bucket_starts_.size() << std::endl;

  return device_vecvec;
fail:
  std::cerr << "Test3.32" << std::endl;
  if (device_bucket_starts) hipFree(device_bucket_starts);
  if (device_data) hipFree(device_data);
  if (device_vecvec) hipFree(device_vecvec);
  return nullptr;
}

template <typename KeyType, typename ValueType>
gpu_vector_map<KeyType, ValueType>* copy_gpu_vector_map_to_device(
    const gpu_vector_map<KeyType, ValueType>* host_vector_map,
    size_t& device_bytes, hipError_t& code) {

  std::cerr << "Test3.12" << std::endl;
  using MapType = gpu_vector_map<KeyType, ValueType>;

  // Schritt 1: Allokieren des Gerätespeichers für die `gpu_vector_map`-Struktur
  MapType* device_vector_map = nullptr;
  ValueType* device_data = nullptr;
  MapType host_map_copy = *host_vector_map;
  CUDA_CALL(hipMalloc(&device_vector_map, sizeof(MapType)));

  // Schritt 2: Allokieren des Gerätespeichers für `el_` (die Daten)
  CUDA_CALL(hipMalloc(&device_data, host_vector_map->size() * sizeof(ValueType)));

  // Schritt 3: Kopieren der Daten vom Host zum Gerät
  CUDA_CALL(hipMemcpy(device_data, host_vector_map->data(),
                    host_vector_map->size() * sizeof(ValueType),
                    hipMemcpyHostToDevice));

  // Schritt 4: Aktualisieren der `el_`-Zeiger im `gpu_vector_map` auf dem Gerät
  host_map_copy.el_ = device_data;

  // Kopieren der aktualisierten `gpu_vector_map`-Struktur zurück auf das Gerät
  CUDA_CALL(hipMemcpy(device_vector_map, &host_map_copy, sizeof(MapType),
                    hipMemcpyHostToDevice));

  hipDeviceSynchronize();
  device_bytes += sizeof(MapType);
  device_bytes += host_vector_map->size() * sizeof(ValueType);
  std::cerr << "Test3.121" << std::endl;
  return device_vector_map;
fail:
    if (device_data) hipFree(device_data);
    if (device_vector_map) hipFree(device_vector_map);
    return nullptr;
}

struct gpu_timetable* create_gpu_timetable(gpu_delta const* route_stop_times,
                                           std::uint32_t  n_route_stop_times,
                                           gpu_vecvec<gpu_route_idx_t,gpu_value_type> const* route_location_seq,
                                           gpu_vecvec<gpu_location_idx_t , gpu_route_idx_t> const* location_routes,
                                           std::uint32_t const* n_locations,
                                           std::uint32_t const* n_routes,
                                           gpu_vector_map<gpu_route_idx_t,gpu_interval<std::uint32_t>> const* route_stop_time_ranges,
                                           gpu_vector_map<gpu_route_idx_t,gpu_interval<gpu_transport_idx_t >> const* route_transport_ranges,
                                           gpu_vector_map<gpu_bitfield_idx_t, gpu_bitfield> const* bitfields,
                                           gpu_vector_map<gpu_transport_idx_t,gpu_bitfield_idx_t> const* transport_traffic_days,
                                           gpu_interval<gpu_sys_days> const* date_range,
                                           gpu_locations const* locations,
                                           gpu_vector_map<gpu_route_idx_t, gpu_clasz> const* route_clasz) {
  size_t device_bytes = 0U;
  std::cerr << "Test3.1" << std::endl;
  hipError_t code;
  gpu_timetable* gtt =
      static_cast<gpu_timetable*>(malloc(sizeof(gpu_timetable)));
  if (gtt == nullptr) {
    printf("nigiri gpu raptor: malloc for gpu_timetable failed\n");
    return nullptr;
  }
  std::cerr << "Test3.2" << std::endl;
  auto locations_copy = *locations;
  // route_stop_times_
  gtt->route_stop_times_ = nullptr;
  CUDA_COPY_TO_DEVICE(gpu_delta, gtt->route_stop_times_, route_stop_times,
                      n_route_stop_times);
  std::cerr << "Test3.24" << std::endl;
  //route_location_seq
  gtt->route_location_seq_ = copy_gpu_vecvec_to_device(route_location_seq,device_bytes,code);


  std::cerr << "Test3.25" << std::endl;
  //location_routes_
  gtt->location_routes_ = copy_gpu_vecvec_to_device(location_routes, device_bytes, code);
  //n_locations_
  std::cerr << "Test3.3" << std::endl;
  gtt->n_locations_ = nullptr;
  CUDA_COPY_TO_DEVICE(uint32_t , gtt->n_locations_, n_locations,1);
  //n_routes_
  gtt->n_routes_ = nullptr;
  CUDA_COPY_TO_DEVICE(uint32_t , gtt->n_routes_, n_routes,1);
  //route_stop_time_ranges_
  gtt->route_stop_time_ranges_ = copy_gpu_vector_map_to_device(route_stop_time_ranges,device_bytes,code);
  //route_transport_ranges_
  gtt->route_transport_ranges_ = copy_gpu_vector_map_to_device(route_transport_ranges,device_bytes,code);
  //bitfields_
  gtt->bitfields_ = copy_gpu_vector_map_to_device(bitfields,device_bytes,code);
  //transport_traffic_days_
  gtt->transport_traffic_days_ = copy_gpu_vector_map_to_device(transport_traffic_days,device_bytes,code);
  //date_range_
  std::cerr << "Test3.4" << std::endl;
  gtt->date_range_ = nullptr;
  using gpu_date_range = gpu_interval<gpu_sys_days>;
  CUDA_COPY_TO_DEVICE(gpu_date_range , gtt->date_range_, date_range,1);
  //locations_
  gtt->locations_ = nullptr;
  locations_copy.transfer_time_ = copy_gpu_vector_map_to_device(locations->transfer_time_,device_bytes,code);
  locations_copy.gpu_footpaths_in_ = copy_gpu_vecvec_to_device(locations->gpu_footpaths_in_,device_bytes,code);
  locations_copy.gpu_footpaths_out_ = copy_gpu_vecvec_to_device(locations->gpu_footpaths_out_,device_bytes,code);
  CUDA_COPY_TO_DEVICE(gpu_locations, gtt->locations_, &locations_copy,1);
  std::cerr << "Test3.5" << std::endl;
  //route_clasz_
  gtt->route_clasz_ = copy_gpu_vector_map_to_device(route_clasz,device_bytes,code);

  hipDeviceSynchronize();
  return gtt;


fail:
  destroy_gpu_timetable(gtt);
  return nullptr;
}
void destroy_gpu_timetable(gpu_timetable* gtt) {
  if (!gtt) return;

  if (gtt->route_stop_times_) hipFree(gtt->route_stop_times_);

  if (gtt->route_location_seq_) {
    if (gtt->route_location_seq_->data_.el_) hipFree(gtt->route_location_seq_->data_.el_);
    if (gtt->route_location_seq_->bucket_starts_.el_) hipFree(gtt->route_location_seq_->bucket_starts_.el_);
    hipFree(gtt->route_location_seq_);
  }

  if (gtt->location_routes_) {
    if (gtt->location_routes_->data_.el_) hipFree(gtt->location_routes_->data_.el_);
    if (gtt->location_routes_->bucket_starts_.el_) hipFree(gtt->location_routes_->bucket_starts_.el_);
    hipFree(gtt->location_routes_);
  }

  if (gtt->n_locations_) hipFree(gtt->n_locations_);
  if (gtt->n_routes_) hipFree(gtt->n_routes_);

  if (gtt->route_stop_time_ranges_) {
    if (gtt->route_stop_time_ranges_->el_) hipFree(gtt->route_stop_time_ranges_->el_);
    hipFree(gtt->route_stop_time_ranges_);
  }

  if (gtt->route_transport_ranges_) {
    if (gtt->route_transport_ranges_->el_) hipFree(gtt->route_transport_ranges_->el_);
    hipFree(gtt->route_transport_ranges_);
  }

  if (gtt->bitfields_) {
    if (gtt->bitfields_->el_) hipFree(gtt->bitfields_->el_);
    hipFree(gtt->bitfields_);
  }

  if (gtt->transport_traffic_days_) {
    if (gtt->transport_traffic_days_->el_) hipFree(gtt->transport_traffic_days_->el_);
    hipFree(gtt->transport_traffic_days_);
  }

  if (gtt->date_range_) hipFree(gtt->date_range_);

  if (gtt->locations_) {
    if (gtt->locations_->transfer_time_) {
      if (gtt->locations_->transfer_time_->el_) hipFree(gtt->locations_->transfer_time_->el_);
      hipFree(gtt->locations_->transfer_time_);
    }
    if (gtt->locations_->gpu_footpaths_in_) {
      if (gtt->locations_->gpu_footpaths_in_->data_.el_) hipFree(gtt->locations_->gpu_footpaths_in_->data_.el_);
      if (gtt->locations_->gpu_footpaths_in_->bucket_starts_.el_) hipFree(gtt->locations_->gpu_footpaths_in_->bucket_starts_.el_);
      hipFree(gtt->locations_->gpu_footpaths_in_);
    }
    if (gtt->locations_->gpu_footpaths_out_) {
      if (gtt->locations_->gpu_footpaths_out_->data_.el_) hipFree(gtt->locations_->gpu_footpaths_out_->data_.el_);
      if (gtt->locations_->gpu_footpaths_out_->bucket_starts_.el_) hipFree(gtt->locations_->gpu_footpaths_out_->bucket_starts_.el_);
      hipFree(gtt->locations_->gpu_footpaths_out_);
    }
    hipFree(gtt->locations_);
  }

  if (gtt->route_clasz_) {
    if (gtt->route_clasz_->el_) hipFree(gtt->route_clasz_->el_);
    hipFree(gtt->route_clasz_);
  }
  free(gtt);
  gtt = nullptr;
  hipDeviceSynchronize();
  auto const last_error = hipGetLastError();
  if (last_error != hipSuccess) {
    printf("CUDA error: %s at " STR(last_error) " %s:%d\n",
           hipGetErrorString(last_error), __FILE__, __LINE__);
  }
}