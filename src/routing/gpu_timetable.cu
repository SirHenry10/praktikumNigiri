
#include "nigiri/routing/gpu_timetable.h"
// todo: types.h include nicht möglich erzeugt hashing error
#include <cstdio>


extern "C" {

#define XSTR(s) STR(s)
#define STR(s) #s

#define CUDA_CALL(call)                                   \
    if ((code = call) != hipSuccess) {                     \
      printf("CUDA error: %s at " STR(call) " %s:%d\n",     \
             hipGetErrorString(code), __FILE__, __LINE__); \
      goto fail;                                            \
    }

#define CUDA_COPY_TO_DEVICE(type, target, source, size)                        \
    CUDA_CALL(hipMalloc(&target, size * sizeof(type)))                          \
    CUDA_CALL(                                                                   \
        hipMemcpy(target, source, size * sizeof(type), hipMemcpyHostToDevice)) \
    device_bytes += size * sizeof(type);

// TODO: const machen alles?
struct gpu_timetable* create_gpu_timetable(gpu_delta const* route_stop_times,
                                             std::uint32_t  n_route_stop_times,
                                             gpu_vecvec<gpu_route_idx_t,gpu_value_type> const* route_location_seq,
                                             gpu_vecvec<gpu_location_idx_t , gpu_route_idx_t> const* location_routes,
                                             std::uint32_t const* n_locations/*,
                                             route_idx_t* location_routes,
                                             std::uint32_t n_locations,
                                             route_idx_t* route_clasz_keys,
                                             clasz* route_clasz_values,
                                             std::uint32_t n_route_clasz,
                                             location_idx_t* transfer_time_keys,
                                             u8_minutes* transfer_time_values,
                                             std::uint32_t n_transfer_time,
                                             footpath* footpaths_out,
                                             std::uint32_t n_footpaths_out,
                                             stop::value_type* route_location_seq,
                                             std::uint32_t n_routes, //= n_route_location_seq
                                             route_idx_t* route_transport_ranges_keys,
                                             interval<transport_idx_t>* route_transport_ranges_values,
                                             std::uint32_t n_route_transport_ranges,
                                             bitfield_idx_t* bitfields_keys,
                                             bitfield* bitfields_values,
                                             std::uint32_t n_bitfields,
                                             transport_idx_t* transport_traffic_days_keys,
                                             bitfield_idx_t* transport_traffic_days_values,
                                             std::uint32_t n_transport_traffic_days,
                                             interval<date::sys_days> date_range,
                                             char* trip_display_names,
                                             std::uint32_t n_trip_display_names,
                                             trip_idx_t* merged_trips,
                                             std::uint32_t n_merged_trips,
                                             merged_trips_idx_t* transport_to_trip_section,
                                             std::uint32_t n_transport_to_trip_section,
                                             transport_idx_t* transport_route_keys,
                                             route_idx_t* transport_route_values,
                                             std::uint32_t n_transport_routes,
                                             route_idx_t* route_stop_time_ranges_keys_keys,
                                             interval<std::uint32_t>* route_stop_time_ranges_values,
                                             std::uint32_t n_route_stop_time_ranges*/) {
  size_t device_bytes = 0U;

  hipError_t code;
  gpu_timetable* gtt =
      static_cast<gpu_timetable*>(malloc(sizeof(gpu_timetable)));
  if (gtt == nullptr) {
    printf("nigiri gpu raptor: malloc for gpu_timetable failed\n");
    return nullptr;
  }
  // route_stop_times_
  gtt->route_stop_times_ = nullptr;
  CUDA_COPY_TO_DEVICE(gpu_delta, gtt->route_stop_times_, route_stop_times,
                      n_route_stop_times);
  //route_location_seq
  gtt->route_location_seq_ = nullptr;
  using gpu_vecvec_route_value = gpu_vecvec<gpu_route_idx_t,gpu_value_type>;
  CUDA_COPY_TO_DEVICE(gpu_vecvec_route_value , gtt->route_location_seq_,
                      route_location_seq, 1);
  //location_routes_
  gtt->location_routes_ = nullptr;
  using gpu_vecvec_location_route = gpu_vecvec<gpu_location_idx_t , gpu_route_idx_t>;
  CUDA_COPY_TO_DEVICE(gpu_vecvec_location_route, gtt->location_routes_, location_routes,1);
  //n_locations_
  gtt->n_locations_ = nullptr;
  CUDA_COPY_TO_DEVICE(uint32_t , gtt->n_locations_, n_locations,1);
  // route_stop_time_ranges
  /*
  gtt->route_stop_time_ranges_keys = nullptr;
  gtt->route_stop_time_ranges_values = nullptr;
  CUDA_COPY_TO_DEVICE(route_idx_t, gtt->route_stop_time_ranges_keys,
  route_stop_time_ranges_keys_keys, n_route_stop_time_ranges);
  CUDA_COPY_TO_DEVICE(interval<std::uint32_t>,
  gtt->route_stop_time_ranges_values, route_stop_time_ranges_values,
                      n_route_stop_time_ranges);
  //route_clasz
  gtt->route_clasz_keys_ = nullptr;
  gtt->route_clasz_values_= nullptr;
  CUDA_COPY_TO_DEVICE(route_idx_t, gtt->route_clasz_keys_, route_clasz_keys,
  n_route_clasz); CUDA_COPY_TO_DEVICE(clasz, gtt->route_clasz_values_,
  route_clasz_values, n_route_clasz);
  //transfer_time_
  gtt->transfer_time_keys_ = nullptr;
  gtt->transfer_time_values_ = nullptr;
  CUDA_COPY_TO_DEVICE(location_idx_t, gtt->transfer_time_keys_,
  transfer_time_keys, n_transfer_time); CUDA_COPY_TO_DEVICE(u8_minutes,
  gtt->transfer_time_values_, transfer_time_values, n_transfer_time);
  //locations_footpaths_out
  gtt->footpaths_out_ = nullptr;
  CUDA_COPY_TO_DEVICE(footpath, gtt->footpaths_out_, footpaths_out,
  n_footpaths_out);
  //route_location_seq
  gtt->route_location_seq_ = nullptr;
  CUDA_COPY_TO_DEVICE(stop::value_type, gtt->route_location_seq_,
  route_location_seq, n_routes);
  //route_transport_ranges
  gtt->route_transport_ranges_keys_ = nullptr;
  gtt->route_transport_ranges_values_ = nullptr;
  CUDA_COPY_TO_DEVICE(route_idx_t, gtt->route_transport_ranges_keys_,
  route_transport_ranges_keys, n_route_transport_ranges);
  CUDA_COPY_TO_DEVICE(interval<transport_idx_t>,
  gtt->route_transport_ranges_values_, route_transport_ranges_values,
  n_route_transport_ranges);
  //bitfields_
  gtt->bitfields_keys_ = nullptr;
  gtt->bitfields_values_ = nullptr;
  CUDA_COPY_TO_DEVICE(bitfield_idx_t, gtt->bitfields_keys_, bitfields_keys,
  n_bitfields); CUDA_COPY_TO_DEVICE(bitfield, gtt->bitfields_values_,
  bitfields_values, n_bitfields);
  //transport_traffic_days_
  gtt->transport_traffic_days_keys_ = nullptr;
  gtt->transport_traffic_days_values_ = nullptr;
  CUDA_COPY_TO_DEVICE(transport_idx_t, gtt->transport_traffic_days_keys_,
  transport_traffic_days_keys, n_transport_traffic_days);
  CUDA_COPY_TO_DEVICE(bitfield_idx_t, gtt->transport_traffic_days_values_,
  transport_traffic_days_values, n_transport_traffic_days);
  //date_range_

  //trip_display_names_
  gtt->trip_display_names_ = nullptr;
  CUDA_COPY_TO_DEVICE(char, gtt->trip_display_names_, trip_display_names,
  n_trip_display_names);
  //merged_trips_
  gtt->merged_trips_ = nullptr;
  CUDA_COPY_TO_DEVICE(trip_idx_t, gtt->merged_trips_, merged_trips,
  n_merged_trips);
  //transport_to_trip_section_
  gtt->transport_to_trip_section_ = nullptr;
  CUDA_COPY_TO_DEVICE(merged_trips_idx_t, gtt->transport_to_trip_section_,
  transport_to_trip_section, n_transport_to_trip_section);
  //transport_route_
  gtt->transport_route_keys_ = nullptr;
  gtt->transport_route_values_ = nullptr;
  CUDA_COPY_TO_DEVICE(transport_idx_t, gtt->transport_route_keys_,
  transport_route_keys, n_transport_routes); CUDA_COPY_TO_DEVICE(route_idx_t,
  gtt->transport_route_values_, transport_route_values, n_transport_routes);
  gtt->n_route_stop_times_ = n_route_stop_times;
  gtt->n_locations_ = n_locations;
  gtt->n_route_clasz_ = n_route_clasz;
  gtt->n_transfer_time_ = n_transfer_time;
  gtt->n_footpaths_out_ = n_footpaths_out;
  gtt->n_routes_ = n_routes;
  gtt->n_route_transport_ranges_ = n_route_transport_ranges;
  gtt->n_bitfields_ = n_bitfields;
  gtt->n_transport_traffic_days_ = n_transport_traffic_days;
  gtt->n_trip_display_names_ = n_trip_display_names;
  gtt->n_merged_trips_ = n_merged_trips;
  gtt->n_transport_to_trip_section_ = n_transport_to_trip_section;
  gtt->n_route_transport_ranges_ = n_transport_routes;
  gtt->n_route_stop_time_ranges_ = n_route_stop_time_ranges;
  gtt->date_range_ = date_range;
   */
  return gtt;

fail:
  // route_stop_times
  hipFree(gtt->route_stop_times_);
  /*
  //route_stop_time_ranges
  hipFree(gtt->route_stop_time_ranges_keys);
  hipFree(gtt->route_stop_time_ranges_values);
  //location_routes_
  hipFree(gtt->location_routes_);
  //route_clasz
  hipFree(gtt->route_clasz_keys_);
  hipFree(gtt->route_clasz_values_);
  //location_transfer_time_count
  hipFree(gtt->transfer_time_keys_);
  hipFree(gtt->transfer_time_values_);
  //locations_footpaths_out
  hipFree(gtt->footpaths_out_);
  //route_location_seq
  hipFree(gtt->route_location_seq_);
  //route_transport_ranges
  hipFree(gtt->route_transport_ranges_keys_);
  hipFree(gtt->route_transport_ranges_values_);
  //bitfields_
  hipFree(gtt->bitfields_keys_);
  hipFree(gtt->bitfields_values_);
  //transport_traffic_days_
  hipFree(gtt->transport_traffic_days_keys_);
  hipFree(gtt->transport_traffic_days_values_);
  //date_range_

  //trip_display_names_
  hipFree(gtt->trip_display_names_);
  //merged_trips_
  hipFree(gtt->merged_trips_);
  //transport_to_trip_section_
  hipFree(gtt->transport_to_trip_section_);
  //transport_route_
  hipFree(gtt->transport_route_keys_);
  hipFree(gtt->transport_route_values_);
  */
  free(gtt);
  return nullptr;
}
void destroy_gpu_timetable(gpu_timetable*& gtt) {
  // route_stop_times_
  hipFree(gtt->route_stop_times_);
  /*
  //route_stop_time_ranges
  hipFree(gtt->route_stop_time_ranges_keys);
  hipFree(gtt->route_stop_time_ranges_values);
  //location_routes_
  hipFree(gtt->location_routes_);
  //route_clasz
  hipFree(gtt->route_clasz_keys_);
  hipFree(gtt->route_clasz_values_);
  //location_transfer_time_count
  hipFree(gtt->transfer_time_keys_);
  hipFree(gtt->transfer_time_values_);
  //locations_footpaths_out
  hipFree(gtt->footpaths_out_);
  //route_location_seq
  hipFree(gtt->route_location_seq_);
  //route_transport_ranges
  hipFree(gtt->route_transport_ranges_keys_);
  hipFree(gtt->route_transport_ranges_values_);
  //bitfields_
  hipFree(gtt->bitfields_keys_);
  hipFree(gtt->bitfields_values_);
  //transport_traffic_days_
  hipFree(gtt->transport_traffic_days_keys_);
  hipFree(gtt->transport_traffic_days_values_);
  //date_range_

  //trip_display_names_
  hipFree(gtt->trip_display_names_);
  //merged_trips_
  hipFree(gtt->merged_trips_);
  //transport_to_trip_section_
  hipFree(gtt->transport_to_trip_section_);
  //transport_route_
  hipFree(gtt->transport_route_keys_);
  hipFree(gtt->transport_route_values_);
  */
  free(gtt);
  gtt = nullptr;
  hipDeviceSynchronize();
  auto const last_error = hipGetLastError();
  if (last_error != hipSuccess) {
    printf("CUDA error: %s at " STR(last_error) " %s:%d\n",
           hipGetErrorString(last_error), __FILE__, __LINE__);
  }
}
}  // extern "C"
//TODO: help methode to convert tt in gtt??