#include "hip/hip_runtime.h"
#include "nigiri/routing/gpu_raptor.cuh"
#include "hip/hip_cooperative_groups.h"


using namespace cooperative_groups;

__device__ __forceinline__ unsigned int get_block_thread_id() {
  return threadIdx.x + (blockDim.x * threadIdx.y);
}

__device__ __forceinline__ unsigned int get_global_thread_id() {
  return get_block_thread_id() + (blockDim.x * blockDim.y * blockIdx.x);
}

__device__ __forceinline__ unsigned int get_block_stride() {
  return blockDim.x * blockDim.y;
}

__device__ __forceinline__ unsigned int get_global_stride() {
  return get_block_stride() * gridDim.x * gridDim.y;
}

// für die uint32_t station/route_marks
__device__ void mark(unsigned int* store, unsigned int const idx) {
  unsigned int const store_idx = (idx >> 5);  // divide by 32
  unsigned int const mask = 1 << (idx % 32);
  atomicOr(&store[store_idx], mask);
}

__device__ bool marked(unsigned int const* const store, unsigned int idx) {
  unsigned int const store_idx = (idx >> 5);  // divide by 32
  unsigned int const val = store[store_idx];
  unsigned int const mask = 1 << (idx % 32);
  return (bool)(val & mask);
}

__device__ void reset_store(unsigned int* store, int const store_size) {
  auto const t_id = get_global_thread_id();
  auto const stride = get_global_stride();

  for (auto idx = t_id; idx < store_size; idx += stride) {
    store[idx] = 0;
  }
}

template <gpu_direction SearchDir, bool Rt>
__device__ void update_time_at_dest(unsigned const k, gpu_delta_t const t, gpu_delta_t* time_at_dest_){
  for (auto i = k; i < nigiri::routing::kMaxTransfers+1; ++i) {
    time_at_dest_[i] = get_best<SearchDir, Rt>(time_at_dest_[i], t);
  }
}

template <gpu_direction SearchDir, bool Rt>
__device__ void convert_station_to_route_marks(unsigned int* station_marks,
                                               unsigned int* route_marks,
                                               bool* any_station_marked,
                                               gpu_raptor<SearchDir, Rt>& gr) {
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  // anstatt stop_count_ brauchen wir location_routes ?location_idx_{gtt.n_locations}?
  for (uint32_t idx = global_t_id; idx < *gr.gtt_.n_locations_; idx += global_stride) {
    if (marked(station_marks, idx)) {
      if (!*any_station_marked) {
        *any_station_marked = true;
      }
      auto const location_routes = gr.gtt_.location_routes_;
      for (auto const& r :  location_routes[gpu_location_idx_t{idx}]) {
        mark(route_marks, gpu_to_idx(r));
      }
      /*if constexpr (Rt) {
        for (auto const& rt_t :
             rtt_->location_rt_transports_[location_idx_t{i}]) {
          any_marked = true;
          state_.rt_transport_mark_[to_idx(rt_t)] = true;
        }
      }*/
    }
  }
}

template <gpu_direction SearchDir, bool Rt>
void reconstruct(nigiri::routing::query const& q, nigiri::routing::journey& j){
  //reconstruct_journey<SearchDir, Rt>(...);
}

template <gpu_direction SearchDir, bool Rt, bool WithClaszFilter>
__device__ bool loop_routes(unsigned const k, gpu_raptor<SearchDir,Rt>& gr){
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  if(get_global_thread_id()==0){
    gr.mem_->device_.any_station_marked_ = false;
  }
  for(auto r_idx = global_t_id; r_idx <= gr.gtt_->n_routes_; r_idx += global_stride){
    auto const r = gpu_route_idx_t{r_idx};
    if(gr.mem_->device_.route_mark_[r_idx]){
      if constexpr (WithClaszFilter){
        if(!is_allowed(gr.allowed_claszes_, gr.gtt_.route_clasz_[r])){
          continue;
        }
      }
      // TODO hier in smaller32 und bigger32 aufteilen? → aber hier geht nur ein thread rein...
      // also sollte vielleicht diese Schleife mit allen auf einmal durchgangen werden???
      gr.mem_->device_.any_station_marked_ |= update_route(k, r, gr);
    }
  }
}

template <gpu_direction SearchDir, bool Rt>
__device__ bool update_route(unsigned const k, gpu_route_idx_t const r, gpu_raptor<SearchDir,Rt>& gr){
  return false;
}

template <gpu_direction SearchDir, bool Rt>
__device__ bool update_route_smaller32(unsigned const k, gpu_raptor<SearchDir,Rt>& gr){
  return false;
}

template <gpu_direction SearchDir, bool Rt>
__device__ bool update_route_bigger32(unsigned const k, gpu_raptor<SearchDir,Rt>& gr){
  return false;
}

template <gpu_direction SearchDir, bool Rt>
__device__ void update_transfers(unsigned const k, gpu_timetable* gtt_, gpu_direction search_dir_,
                                 bool* is_dest_, uint16_t* dist_to_end_, gpu_delta_t* tmp_,
                                 gpu_delta_t* best_, gpu_delta_t* time_at_dest_,
                                 uint16_t* lb_, gpu_delta_t* round_times_, uint32_t row_count_round_times_,
                                 uint32_t* station_mark_, uint32_t* prev_station_mark_){
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  for(auto l_idx = global_t_id;
       reinterpret_cast<uint32_t*>(l_idx) <= gtt_->n_locations_; l_idx += global_stride){
    if(!prev_station_mark_[l_idx]){ //PREV_Stations????
      continue;
    }
    auto const is_dest = is_dest_[l_idx];
    // wo sind unsere locations bzw.wie heißen sie?
    //TODO locations in gtt hinzufügen
    auto const transfer_time = (dist_to_end_.empty() && is_dest)
        ? 0 : dir<search_dir_, Rt>(gtt_->locations_.transfer_time_[gpu_location_idx_t{l_idx}]).count();
    auto const fp_target_time =
        static_cast<gpu_delta>(tmp_[l_idx] + transfer_time);
    if(is_better(fp_target_time, best_[l_idx])
        && is_better(fp_target_time, time_at_dest_[k])){
      if(lb_[l_idx] == kUnreachable_
          || !is_better(fp_target_time + dir<search_dir_, Rt>(lb_[l_idx]), time_at_dest_[k])){
        continue;
      }

      round_times_[k * row_count_round_times_ + l_idx] = fp_target_time;
      best_[l_idx] = fp_target_time;
      station_mark_[l_idx] = true;
      if(is_dest){
        update_time_at_dest(k, fp_target_time, time_at_dest_);
      }
    }
  }
}

template <gpu_direction SearchDir, bool Rt>
__device__ void update_footpaths(unsigned const k, gpu_profile_idx_t const prf_idx,
                                 uint32_t* prev_station_mark_, gpu_direction search_dir_,
                                 gpu_timetable* gtt_, gpu_delta_t* tmp_, gpu_delta_t* best_,
                                 uint16_t* lb_, uint32_t* station_mark_, gpu_delta_t* time_at_dest_,
                                 bool* is_dest_, gpu_delta_t* round_times_, uint32_t row_count_round_times_){
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  for(auto idx = global_t_id;
       reinterpret_cast<uint32_t*>(idx) <= gtt_->n_locations_; idx += global_stride){
    if(!prev_station_mark_[idx]){//PREV_Stations??
      continue;
    }
    auto const l_idx = gpu_location_idx_t{idx};
    auto const& fps = (search_dir_ == gpu_direction::kForward)
         ? gtt_->locations_.footpaths_out_[prf_idx][l_idx] : gtt_->locations_.footpaths_in_[prf_idx][l_idx];
    for(auto const& fp: fps){
      auto const target = to_idx(fp.target());
      auto const fp_target_time =
          gpu_clamp(tmp_[idx] + dir<search_dir_, Rt>(fp.duration()).count());

      if(is_better(fp_target_time, best_[target])
          && is_better(fp_target_time, time_at_dest_[k])){
        auto const lower_bound = lb_[to_idx(fp.target())];
        if(lower_bound == kUnreachable_
            || !is_better(fp_target_time + dir<search_dir_, Rt>(lower_bound), time_at_dest_[k])){
          continue;
        }
      }
      round_times_[k * row_count_round_times_ + to_idx(fp.target())] = fp_target_time;
      best_[to_idx(fp.target())] = fp_target_time;
      station_mark_[to_idx(fp.target())] = true;
      if(is_dest_[to_idx(fp.target())]){
        update_time_at_dest(k, fp_target_time, gr);
      }
    }
  }

}

template <gpu_direction SearchDir, bool Rt>
__device__ void update_intermodal_footpaths(unsigned const k, gpu_raptor<SearchDir,Rt>& gr){
  if(get_global_thread_id()==0 && gr.dist_to_end_.empty()){
    return;
  }
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  for(auto idx = global_t_id; idx <= gr.gtt_.n_locations_; idx += global_stride){
    if((gr.mem_->device_ || gr.mem_->device_.station_mark_[idx]) && gr.dist_to_end_[idx] != gr.kUnreachable){
      auto const end_time = clamp(gr.get_best(gr.mem_->device_.best_[idx], gr.mem_->device_.tmp_[idx]) + gr.dir(gr.dist_to_end_[idx]));
      if(gr.is_better(end_time, gr.mem_->device_.best_[gr.kIntermodalTarget])){
        gr.mem_->device_.round_times_[k][gr.kIntermodalTarget] = end_time;
        gr.mem_->device_.best_[gr.IntermodalTarget] = end_time;
        update_time_at_dest(k, end_time, gr);
      }
    }
  }
}

template <gpu_direction SearchDir, bool Rt>
__device__ gpu_transport get_earliest_transport(unsigned const k,
                                       gpu_route_idx_t const r,
                                       gpu_stop_idx_t const stop_idx,
                                       gpu_day_idx_t const day_at_stop,
                                       gpu_minutes_after_midnight_t const mam_at_stop,
                                       gpu_location_idx_t const l,
                                       gpu_raptor<SearchDir,Rt>& gr){
  auto const n_days_to_iterate = std::min(gr.gtt_.kMaxTravelTime.count()/1440 +1,
                                          gr.kFwd ? gr.n_days_ - gr.as_int(day_at_stop) : gr.as_int(day_at_stop)+1);
  auto const event_times =
      gr.gtt_.event_times_at_stop(r, stop_idx, gr.kFwd ? gpu_event_type::kDep : gpu_event_type::kArr);
  auto const seek_first_day = [&]() {
    return linear_lb(get_begin_it(event_times), get_end_it(event_times), mam_at_stop,
                     [&](gpu_delta const a, gpu_minutes_after_midnight_t const b) {
                       return gr.is_better(a.mam_, b.count()); // anders mit gpu_delta umgehen
                     });
  };

  // for Schleife über n_days_to_iterate
}

template <gpu_direction SearchDir, bool Rt>
__device__ bool is_transport_active(gpu_transport_idx_t const t, std::size_t const day , gpu_timetable* gtt_)  {
  return gtt_->bitfields_[gtt_->transport_traffic_days_[t]].test(day);
}

template <gpu_direction SearchDir, bool Rt>
__device__ gpu_delta_t time_at_stop(gpu_route_idx_t const r, gpu_transport const t,
                                    gpu_stop_idx_t const stop_idx, gpu_event_type const ev_type,
                                    gpu_timetable* gtt_, gpu_strong<uint16_t, _day_idx> base_){
  auto const range = *gtt_->route_transport_ranges_;
  auto const n_transports = static_cast<unsigned>(range.size());
  auto const route_stop_begin = static_cast<unsigned>(range[r].from_.v_ + n_transports *
                                (stop_idx * 2 - (ev_type==gpu_event_type::kArr ? 1 : 0)));
  return gpu_clamp((as_int(t.day_) - as_int(base_)) * 1440
                   + gtt_->route_stop_times_[route_stop_begin + (gpu_to_idx(t.day_) - gpu_to_idx(range[r].from_))].count());
}

template <gpu_direction SearchDir, bool Rt>
__device__ void raptor_round(unsigned const k, gpu_profile_idx_t const prf_idx, gpu_timetable* gtt_,
                             gpu_strong<uint16_t, _day_idx> base_, gpu_clasz_mask_t allowed_claszes_, uint16_t* dist_to_end_,
                             bool* is_dest_, uint16_t* lb_, int n_days_, gpu_direction search_dir_, gpu_delta_t* time_at_dest_,
                             bool any_station_marked_, uint32_t* route_mark_, uint32_t* station_mark_, gpu_delta_t* best_,
                             gpu_delta_t* round_times_, gpu_delta_t* tmp, uint32_t size_best_, uint32_t size_tmp_,
                             uint32_t row_count_round_times_, uint32_t column_count_round_times_,
                             uint32_t size_route_mark_, uint32_t size_station_mark_){

  // update_time_at_dest für alle locations
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  //TODO sicher, dass man über n_locations iterieren muss? -> aufpassen, dass round_times nicht out of range zugegriffen wird
  for(auto idx = global_t_id; idx < *gtt_->n_locations_; idx += global_stride){
    best_[global_t_id] = get_best<search_dir_, Rt>(round_times_[k*row_count_round_times_+idx], best_[idx]);
    if(is_dest_[idx]){
      update_time_at_dest<search_dir_, Rt>(k, best_[global_t_id], time_at_dest_);
    }
  }
  this_grid().sync();

  // für jede location & für jede location_route state_.route_mark_
  if(get_global_thread_id()==0){
    any_station_marked_ = false;
  }
  convert_station_to_route_marks<search_dir_, Rt>(station_mark_, route_mark_, any_station_marked_, gtt_);
  this_grid().sync();

  if(get_global_thread_id()==0){
    if(!any_station_marked_){
      return;
    }
    // SWAP ???
    utl::fill(station_mark_, false); //TODO fill() auf dem device nutzbar machen
  }
  this_grid().sync();
  // loop_routes mit true oder false
  // any_station_marked soll nur einmal gesetzt werden, aber loop_routes soll mit allen threads durchlaufen werden?
  any_station_marked_ = (allowed_claszes_ = nigiri::routing::all_clasz_allowed())
                                              ? loop_routes<false>(k, gr) : loop_routes<false>(k, gr);
  this_grid().sync();
  if(get_global_thread_id()==0){
    if(!any_station_marked_){
      return;
    }
    utl::fill(route_mark_, false);
    // SWAP ???
    utl::fill(station_mark_, false);
  }
  this_grid().sync();

  // update_transfers
  update_transfers(k, gr);
  this_grid().sync();
  // update_footpaths
  update_footpaths(k, prf_idx, gr); //müssen wir gtt hier mit einsetzen??
  this_grid().sync();
  // update_intermodal_footpaths
  update_intermodal_footpaths(k, gr);

}

template <gpu_direction SearchDir, bool Rt>
__device__ void init_arrivals(gpu_delta_t d_worst_at_dest, gpu_unixtime_t const worst_time_at_dest, gpu_day_idx_t* base_, gpu_delta_t* time_at_dest, gpu_timetable* gtt_){
  auto const t_id = get_global_thread_id();

  if(t_id==0){
    d_worst_at_dest = unix_to_gpu_delta(base(gtt_, base_), worst_time_at_dest);
  }

  if(t_id < nigiri::routing::kMaxTransfers+1){
    time_at_dest[t_id] = get_best<SearchDir>(d_worst_at_dest, time_at_dest[t_id]);
  }

}

// größten Teil von raptor.execute() wird hierdrin ausgeführt
// kernel muss sich außerhalb der gpu_raptor Klasse befinden
template <gpu_direction SearchDir, bool Rt>
__global__ void gpu_raptor_kernel(gpu_unixtime_t const start_time,
                                  uint8_t const max_transfers,
                                  gpu_unixtime_t const worst_time_at_dest,
                                  gpu_profile_idx_t const prf_idx,
                                  nigiri::pareto_set<nigiri::routing::journey>& results,
                                  gpu_raptor<SearchDir,Rt>& gr){
  auto const end_k = std::min(max_transfers, nigiri::routing::kMaxTransfers) + 1U;
  // 1. Initialisierung
  gpu_delta_t d_worst_at_dest{};
  init_arrivals(d_worst_at_dest, worst_time_at_dest, gr.base_, gr.mem_->device_.time_at_dest_, gr.gtt_);
  this_grid().sync();

  // 2. Update Routes
  for (auto k = 1U; k != end_k; ++k) { // diese Schleife bleibt, da alle Threads in jede Runde gehen

    // Resultate aus lezter Runde von device in variable speichern?
    raptor_round(k, prf_idx, gr.gtt_, *gr.base_, *gr.allowed_claszes_,
                 gr.dist_to_end_, gr.is_dest_, gr.lb_, *gr.n_days_, gr.mem_->device_.search_dir_,
                 gr.mem_->device_.time_at_dest_, *gr.mem_->device_.any_station_marked_,
                 gr.mem_->device_.route_mark_, gr.mem_->device_.station_mark_,
                 gr.mem_->device_.best_, gr.mem_->device_.round_times_, gr.mem_->device_.tmp_,
                 gr.mem_->device_.size_best_, gr.mem_->device_.size_tmp_,
                 gr.mem_->device_.row_count_round_times_, gr.mem_->device_.column_count_round_times_,
                 gr.mem_->device_.size_route_mark_, gr.mem_->device_.size_station_mark_,
                 gr.kUnreachable, gr.kIntermodalTarget);
    this_grid().sync();
  }
  this_grid().sync();

  //construct journey

  this_grid().sync();

}


