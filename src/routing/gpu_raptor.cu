#include "hip/hip_runtime.h"
#pragma once
#include "nigiri/routing/gpu_raptor.h"
#include <iostream>

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;
// leader type must be unsigned 32bit
// no leader is a zero ballot vote (all 0) minus 1 => with underflow all 1's
constexpr unsigned int FULL_MASK = 0xFFFFffff;
constexpr unsigned int NO_LEADER = FULL_MASK;

__device__ __forceinline__ unsigned int get_block_thread_id() {
  return threadIdx.x + (blockDim.x * threadIdx.y);
}

__device__ __forceinline__ unsigned int get_global_thread_id() {
  return get_block_thread_id() + (blockDim.x * blockDim.y * blockIdx.x);
}

__device__ __forceinline__ unsigned int get_block_stride() {
  return blockDim.x * blockDim.y;
}

__device__ __forceinline__ unsigned int get_global_stride() {
  return get_block_stride() * gridDim.x * gridDim.y;
}

// für die uint32_t station/route_marks
__device__ void mark(unsigned int* store, unsigned int const idx) {
  unsigned int const store_idx = (idx >> 5);  // divide by 32
  unsigned int const mask = 1 << (idx % 32);
  atomicOr(&store[store_idx], mask);
}

__device__ bool marked(unsigned int const* const store, unsigned int idx) {
  unsigned int const store_idx = (idx >> 5);  // divide by 32
  unsigned int const val = store[store_idx];
  unsigned int const mask = 1 << (idx % 32);
  return (bool)(val & mask);
}

__device__ void reset_store(unsigned int* store, int const store_size) {
  auto const t_id = get_global_thread_id();
  auto const stride = get_global_stride();

  for (auto idx = t_id; idx < store_size; idx += stride) {
    store[idx] = 0x000;
  }
}
__device__ void swap_b_reset(unsigned int* store_a, unsigned int* store_b, int const store_size) {
  auto const t_id = get_global_thread_id();
  auto const stride = get_global_stride();
  for (auto idx = t_id; idx < store_size; idx += stride) {
    store_a[idx] = store_b[idx];
    store_b[idx] = 0;
  }
}

template <gpu_direction SearchDir>
__device__ bool update_arrival(gpu_delta_t* base_,
                               const unsigned int l_idx, gpu_delta_t const val){
  gpu_delta_t* const arr_address = &base_[l_idx];
  auto* base_address = (int*)((size_t)arr_address & ~2);  // Verwende `int*` für vorzeichenbehaftete Werte
  int old_value, new_value;

  do {
    // Lese den gesamten 32-Bit-Wert atomar
    old_value = atomicCAS(base_address, *base_address, *base_address);

    // Setze den 16-Bit-Wert korrekt in die oberen oder unteren 16 Bit
    if ((size_t)arr_address & 2) {
      // Wir arbeiten an den oberen 16 Bits
      int old_upper = (old_value >> 16) & 0xFFFF;
      // Korrektur für negative Werte
      old_upper = (old_upper << 16) >> 16;
      int new_upper = get_best<SearchDir>(old_upper, val);
      if (new_upper == old_upper) {
        return false;
      }
      new_value = (old_value & 0x0000FFFF) | (new_upper << 16);
    } else {
      // Wir arbeiten an den unteren 16 Bits
      int old_lower = old_value & 0xFFFF;
      // Korrektur für negative Werte
      old_lower = (old_lower << 16) >> 16;
      int new_lower = get_best<SearchDir>(old_lower, val);
      if (new_lower == old_lower){
        return false;
      }
      new_value = (old_value & 0xFFFF0000) | (new_lower & 0xFFFF);
    }
    // Versuche, den neuen 32-Bit-Wert atomar zu schreiben, falls sich der Wert geändert hat
  } while (atomicCAS(base_address, old_value, new_value) != old_value);

  return true;
}

template <gpu_direction SearchDir, bool Rt>
__device__ void update_time_at_dest(unsigned const k, gpu_delta_t const t, gpu_delta_t * time_at_dest_){
  for (auto i = k; i < gpu_kMaxTransfers+1; ++i) {
    time_at_dest_[i] = get_best<SearchDir>(time_at_dest_[i], t);
  }
}

template <gpu_direction SearchDir, bool Rt>
__device__ void convert_station_to_route_marks(unsigned int* station_marks, unsigned int* route_marks,
                                               int* any_station_marked,
                                               gpu_vecvec<gpu_location_idx_t , gpu_route_idx_t> const* location_routes_,
                                               std::uint32_t const n_locations) {
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  // anstatt stop_count_ brauchen wir location_routes ?location_idx_{n_locations}?
  for (uint32_t idx = global_t_id; idx < n_locations; idx += global_stride) {
    if (marked(station_marks, idx)) {
      if (!*any_station_marked) {
        atomicOr(reinterpret_cast<int*>(any_station_marked),1);
      }
      for (auto r : (*location_routes_)[gpu_location_idx_t{idx}]) {

        mark(route_marks, gpu_to_idx(r));
      }
    }
  }
}

template <gpu_direction SearchDir, bool Rt>
__device__ gpu_delta_t time_at_stop(gpu_route_idx_t const r, gpu_transport const t,
                                    gpu_stop_idx_t const stop_idx,
                                    gpu_event_type const ev_type,
                                    gpu_day_idx_t base_,
                                    gpu_vector_map<gpu_route_idx_t,gpu_interval<std::uint32_t>> const* route_stop_time_ranges,
                                    gpu_vector_map<gpu_route_idx_t,gpu_interval<gpu_transport_idx_t >> const* route_transport_ranges,
                                    gpu_delta const* route_stop_times){
  auto const n_transports = static_cast<unsigned>((*route_transport_ranges)[r].size());
  auto const route_stop_begin = static_cast<unsigned>( (*route_stop_time_ranges)[r].from_ + n_transports *
                                                                              (stop_idx * 2 - (ev_type==gpu_event_type::kArr ? 1 : 0)));
  return gpu_clamp((as_int(t.day_) - as_int(base_)) * 1440
                   + route_stop_times[route_stop_begin +
                                             (gpu_to_idx(t.t_idx_) - gpu_to_idx((*route_transport_ranges)[r].from_))].count());
}

template <typename It, typename End, typename Key, typename Cmp>
__device__ It linear_lb(It from, End to, Key&& key, Cmp&& cmp) {
  for (auto it = from; it != to; ++it) {
    if (!cmp(*it, key)) {
      return it;
    }
  }
  return to;
}

template <gpu_direction SearchDir, bool Rt>
__device__ bool is_transport_active(gpu_transport_idx_t const t,
                                    std::size_t const day,
                                    gpu_vector_map<gpu_transport_idx_t,gpu_bitfield_idx_t> const* transport_traffic_days,
                                    gpu_vector_map<gpu_bitfield_idx_t, gpu_bitfield> const* bitfields)  {
  assert((*transport_traffic_days).el_ != nullptr);
  assert((*bitfields).el_ !=  nullptr);
  if (day >= (*bitfields)[(*transport_traffic_days)[t]].size()) {
    return false;
  }
  auto const block = (*bitfields)[(*transport_traffic_days)[t]].blocks_[day / (*bitfields)[(*transport_traffic_days)[t]].bits_per_block];
  auto const bit = (day % (*bitfields)[(*transport_traffic_days)[t]].bits_per_block);
  return (block & (std::uint64_t{1U} << bit)) != 0U;
}

template <gpu_direction SearchDir>
__device__ bool valid(gpu_delta_t t) {
  // Use if constexpr to ensure compile-time evaluation
  if constexpr (SearchDir == gpu_direction::kForward) {
    return t != cuda::std::numeric_limits<gpu_delta_t>::max();
  } else {
    return t != cuda::std::numeric_limits<gpu_delta_t>::min();
  }
}

//hilfsmethode für update_route
template <gpu_direction SearchDir, bool Rt>
__device__ gpu_transport get_earliest_transport(unsigned const k,
                                                gpu_route_idx_t const r,
                                                gpu_stop_idx_t const stop_idx,
                                                gpu_day_idx_t const day_at_stop,
                                                gpu_minutes_after_midnight_t const mam_at_stop,
                                                gpu_location_idx_t const l,
                                                gpu_raptor_stats* stats_,
                                                uint16_t* lb_,
                                                gpu_delta_t* time_at_dest_,
                                                gpu_vector_map<gpu_route_idx_t,gpu_interval<gpu_transport_idx_t >> const* route_transport_ranges,
                                                gpu_vector_map<gpu_route_idx_t,gpu_interval<uint32_t >> const* route_stop_time_ranges,
                                                int n_days_, gpu_day_idx_t* base_,
                                                gpu_delta const* route_stop_times,
                                                gpu_vector_map<gpu_bitfield_idx_t, gpu_bitfield> const* bitfields,
                                                gpu_vector_map<gpu_transport_idx_t,gpu_bitfield_idx_t> const* transport_traffic_days) {
  ++stats_[get_global_thread_id()%32].n_earliest_trip_calls_;

  auto const n_days_to_iterate = get_smaller(
      gpu_kMaxTravelTime.count() / 1440 + 1,
      (SearchDir == gpu_direction::kForward) ? n_days_ - as_int(day_at_stop) : as_int(day_at_stop) + 1);

  auto const event_times = gpu_event_times_at_stop(
      r, stop_idx, (SearchDir == gpu_direction::kForward) ? gpu_event_type::kDep : gpu_event_type::kArr, route_stop_time_ranges,route_transport_ranges, route_stop_times);


  auto const seek_first_day = [&]() {
    return linear_lb(gpu_get_begin_it<SearchDir>(event_times), gpu_get_end_it<SearchDir>(event_times),
                     mam_at_stop,
                     [&](gpu_delta const a, gpu_minutes_after_midnight_t const b) {
                       return is_better<SearchDir>(a.mam_, b.count());
                     });
  };

#if defined(NIGIRI_TRACING)
  auto const l_idx =
      stop{tt_.route_location_seq_[r][stop_idx]}.location_idx();

  trace(
      "┊ │k={}    et: current_best_at_stop={}, stop_idx={}, location={}, "
      "n_days_to_iterate={}\n",
      k, tt_.to_unixtime(day_at_stop, mam_at_stop), stop_idx,
      location{tt_, l_idx}, n_days_to_iterate);
#endif

  for (auto i = gpu_day_idx_t::value_t{0U}; i != n_days_to_iterate; ++i) {
    auto const ev_time_range =
        gpu_it_range{i == 0U ? seek_first_day() : gpu_get_begin_it<SearchDir>(event_times),
                 gpu_get_end_it<SearchDir>(event_times)};

    if (ev_time_range.empty()) {
      continue;
    }

    auto const day = (SearchDir == gpu_direction::kForward) ? day_at_stop + i : day_at_stop - i;
    for (auto it = begin(ev_time_range); it != end(ev_time_range); ++it) {
      auto const t_offset =
          static_cast<std::size_t>(&*it - event_times.data());
      auto const ev = *it;
      auto const ev_mam = ev.mam_;

      if (is_better_or_eq<SearchDir>(time_at_dest_[k],
                          to_gpu_delta(day, ev_mam, base_) + dir<SearchDir>(lb_[gpu_to_idx(l)]))) {
        //hier geht cpu rein
        return {gpu_transport_idx_t::invalid(), gpu_day_idx_t::invalid()};
      }
      auto const t = (*route_transport_ranges)[r][t_offset];
      if (i == 0U && !is_better_or_eq<SearchDir>(mam_at_stop.count(), ev_mam)) {
        continue;
      }

      auto const ev_day_offset = ev.days_;
      auto const start_day =
          static_cast<std::size_t>(as_int(day) - ev_day_offset);

      if(!is_transport_active<SearchDir, Rt>(t, start_day, transport_traffic_days, bitfields)) {
        continue;
      }
      return {t, static_cast<gpu_day_idx_t>(as_int(day) - ev_day_offset)};
    }
  }
  return {};
}

//nicht parallele update_route
template <gpu_direction SearchDir, bool Rt>
__device__ void update_route(unsigned const k, gpu_route_idx_t const r,
                  gpu_vecvec<gpu_route_idx_t,gpu_value_type> const* route_location_seq,
                  gpu_raptor_stats* stats_,
                  uint32_t* prev_station_mark_, gpu_delta_t* best_,
                  gpu_delta_t* round_times_, uint32_t column_count_round_times_,
                  gpu_delta_t* tmp_,
                  uint16_t* lb_,
                  gpu_delta_t* time_at_dest_,
                  uint32_t* station_mark_,
                  unsigned short kUnreachable,
                  int* any_station_marked_,
                  gpu_day_idx_t* base_,
                             gpu_vector_map<gpu_route_idx_t,gpu_interval<gpu_transport_idx_t >> const* route_transport_ranges,
                  gpu_vector_map<gpu_route_idx_t,gpu_interval<uint32_t>> const* route_stop_time_ranges,
                  int n_days_, gpu_vector_map<gpu_bitfield_idx_t, gpu_bitfield> const* bitfields,
                  gpu_delta const* route_stop_times,
                  gpu_vector_map<gpu_transport_idx_t,gpu_bitfield_idx_t> const* transport_traffic_days) {
  auto const stop_seq = (*route_location_seq)[r];
  // diese Variable ist das Problem beim Parallelisieren
  auto et = gpu_transport{};
  // hier gehen wir durch alle Stops der Route r → das wollen wir in update_smaller/bigger machen
  for (auto i = 0U; i != stop_seq.size(); ++i) {
    auto const stop_idx =
        static_cast<gpu_stop_idx_t>((SearchDir == gpu_direction::kForward) ? i : stop_seq.size() - i - 1U);
    auto const stp = gpu_stop{stop_seq[stop_idx]};
    auto const l_idx = gpu_to_idx(stp.gpu_location_idx());
    auto const is_last = i == stop_seq.size() - 1U;

    // wenn transportmittel an dem Tag nicht fährt &
    // wenn station nicht markiert ist, wird diese übersprungen → springt zur nächsten station
    if (!et.is_valid() && !marked(prev_station_mark_, l_idx)) {
      continue;
    }
    auto current_best = kInvalidGpuDelta<SearchDir>;
    //wenn station ausgehende/eingehende Transportmittel hat & transportmittel an dem Tag fährt

    if (et.is_valid() && ((SearchDir == gpu_direction::kForward) ? stp.out_allowed() : stp.in_allowed())) {
      // wann transportmittel an dieser station ankommt
      auto const by_transport = time_at_stop<SearchDir, Rt>(
          r, et, stop_idx, (SearchDir == gpu_direction::kForward) ? gpu_event_type::kArr : gpu_event_type::kDep, *base_, route_stop_time_ranges, route_transport_ranges, route_stop_times);
      // beste Zeit für diese station bekommen
      current_best = get_best<SearchDir>(round_times_[(k - 1)*column_count_round_times_ + l_idx],
                              tmp_[l_idx], best_[l_idx]);
      assert(by_transport != cuda::std::numeric_limits<gpu_delta_t>::min() &&
             by_transport != cuda::std::numeric_limits<gpu_delta_t>::max());
      // wenn Ankunftszeit dieses Transportmittels besser ist als beste Ankunftszeit für station
      // & vor frühster Ankunftszeit am Ziel liegt
      if (is_better<SearchDir>(by_transport, current_best) &&
          is_better<SearchDir>(by_transport, time_at_dest_[k]) &&
          lb_[l_idx] != kUnreachable &&
          is_better<SearchDir>(by_transport + dir<SearchDir>(lb_[l_idx]), time_at_dest_[k])) {
        // dann wird frühste Ankunftszeit an dieser Station aktualisiert
        // hier einziger Punkt, wo gemeinsame Variablen verändert werden → ATOMIC
        auto updated = update_arrival<SearchDir>(tmp_,l_idx,get_best<SearchDir>(by_transport, tmp_[l_idx]));
        if (updated){
          ++stats_[get_global_thread_id()%32].n_earliest_arrival_updated_by_route_;
          mark(station_mark_, l_idx);
          current_best = by_transport;
          atomicOr(any_station_marked_,1);
        }
      }
    }

    // wenn es die letzte Station in der Route ist
    // oder es keine ausgehenden/eingehenden transportmittel gibt
    // oder die Station nicht markiert war

    if (is_last || !((SearchDir == gpu_direction::kForward) ? stp.in_allowed() : stp.out_allowed()) ||
        !marked(prev_station_mark_, l_idx)) {
      //dann wird diese übersprungen
      continue;
    }

    // wenn der lowerBound von der Station nicht erreichbar ist,
    // werden die darauffolgenden Stationen auch nicht erreichbar sein

    if (lb_[l_idx] == kUnreachable) {
      // dann wird Durchgehen dieser Route abgebrochen
      break;
    }

    // wenn Transportmittel an dem Tag fährt, dann ist das hier Ankunftszeit am Stop
    auto const et_time_at_stop =
        et.is_valid()
            ? time_at_stop<SearchDir, Rt>(r, et, stop_idx,
                           (SearchDir == gpu_direction::kForward) ? gpu_event_type::kDep : gpu_event_type::kArr,
                           *base_, route_stop_time_ranges, route_transport_ranges, route_stop_times)
            : kInvalidGpuDelta<SearchDir>;
    // vorherige Ankunftszeit an der Station
    auto const prev_round_time = round_times_[(k-1) * column_count_round_times_ + l_idx];

    assert(prev_round_time != kInvalidGpuDelta<SearchDir>);
    // wenn vorherige Ankunftszeit besser ist → dann sucht man weiter nach besserem Umstieg in ein Transportmittel

    if (is_better_or_eq<SearchDir>(prev_round_time, et_time_at_stop)) {

      auto const [day, mam] = gpu_split_day_mam(*base_, prev_round_time);
      // Hier muss leader election stattfinden
      // dann wird neues Transportmittel, das am frühsten von station abfährt

      auto const new_et = get_earliest_transport<SearchDir, Rt>(k, r, stop_idx, day, mam,
                                                 stp.gpu_location_idx(), stats_, lb_, time_at_dest_,route_transport_ranges,
                                                                route_stop_time_ranges, n_days_, base_, route_stop_times, bitfields, transport_traffic_days);
      current_best =
          get_best<SearchDir>(current_best, best_[l_idx], tmp_[l_idx]);
      // wenn neues Transportmittel an diesem Tag fährt und
      // bisherige beste Ankunftszeit Invalid ist ODER Ankunftszeit an Station besser als Ankunftszeit von neuem Transportmittel
      if (new_et.is_valid() &&
          (current_best == kInvalidGpuDelta<SearchDir> ||
           is_better_or_eq<SearchDir>(
               time_at_stop<SearchDir, Rt>(r, new_et, stop_idx,
                            (SearchDir == gpu_direction::kForward) ? gpu_event_type::kDep : gpu_event_type::kArr, *base_, route_stop_time_ranges, route_transport_ranges, route_stop_times),
               et_time_at_stop))) {
        // dann wird neues Transportmittel genommen
        et = new_et;
      }
    }
  }
}

template <gpu_direction SearchDir, bool Rt, bool WithClaszFilter>
__device__ void loop_routes(unsigned const k, int* any_station_marked_, uint32_t* route_mark_,
                            gpu_clasz_mask_t const* allowed_claszes_,
                            gpu_raptor_stats* stats_,
                            short const* kMaxTravelTimeTicks_, uint32_t* prev_station_mark_,
                            gpu_delta_t* best_,
                            gpu_delta_t* round_times_, uint32_t column_count_round_times_,
                            gpu_delta_t* tmp_,
                            uint16_t* lb_, int n_days_,
                            gpu_delta_t* time_at_dest_,
                            uint32_t* station_mark_, gpu_day_idx_t* base_,
                            unsigned short kUnreachable,
                            gpu_delta const* route_stop_times,
                            gpu_vecvec<gpu_route_idx_t,gpu_value_type> const* route_location_seq,
                            gpu_vecvec<gpu_location_idx_t , gpu_route_idx_t> const* location_routes,
                            std::uint32_t const n_locations,
                            std::uint32_t const n_routes,
                            gpu_vector_map<gpu_route_idx_t,gpu_interval<std::uint32_t>> const* route_stop_time_ranges,
                            gpu_vector_map<gpu_route_idx_t,gpu_interval<gpu_transport_idx_t >> const* route_transport_ranges,
                            gpu_vector_map<gpu_bitfield_idx_t, gpu_bitfield> const* bitfields,
                            gpu_vector_map<gpu_transport_idx_t,gpu_bitfield_idx_t> const* transport_traffic_days,
                            gpu_interval<gpu_sys_days> const* date_range,
                            gpu_vector_map<gpu_location_idx_t, gpu_u8_minutes> const* transfer_time,
                            gpu_vecvec<gpu_location_idx_t, nigiri::gpu_footpath> const* gpu_footpaths_out,
                            gpu_vecvec<gpu_location_idx_t, nigiri::gpu_footpath> const* gpu_footpaths_in,
                            gpu_vector_map<gpu_route_idx_t, gpu_clasz> const* route_clasz){
  if(get_global_thread_id() == 0){
    atomicAnd(any_station_marked_,0);
  }
  this_grid().sync();
  auto const global_t_id = get_global_thread_id();
  auto const stride = blockDim.y * gridDim.x;
  auto const start_r_id = threadIdx.y + (blockDim.y * blockIdx.x);
  for(auto r_idx = start_r_id;
       r_idx < n_routes; r_idx += stride){
    auto const r = gpu_route_idx_t{r_idx};
    if(!marked(route_mark_, r_idx)) {
      continue;
    }
    if constexpr (WithClaszFilter){
      auto const as_mask = static_cast<gpu_clasz_mask_t>(1U << static_cast<std::underlying_type_t<gpu_clasz>>((*route_clasz)[r]));
      if(!((*allowed_claszes_ & as_mask)==as_mask)){
        continue;
      }
    }
    ++stats_[global_t_id%32].n_routes_visited_;
    update_route<SearchDir, Rt>(k, r, route_location_seq, stats_, prev_station_mark_, best_, round_times_,
                                column_count_round_times_, tmp_, lb_, time_at_dest_, station_mark_, kUnreachable, any_station_marked_,
                                base_,route_transport_ranges,route_stop_time_ranges, n_days_, bitfields, route_stop_times, transport_traffic_days);
  }
}

template <gpu_direction SearchDir, bool Rt>
__device__ void update_transfers(unsigned const k, bool const * is_dest_, uint16_t* dist_to_end_,
                                 uint32_t dist_to_end_size_, gpu_delta_t* tmp_,
                                 gpu_delta_t* best_, gpu_delta_t* time_at_dest_, unsigned short kUnreachable,
                                 uint16_t* lb_, gpu_delta_t* round_times_, uint32_t column_count_round_times_,
                                 uint32_t* station_mark_, uint32_t* prev_station_mark_,
                                 std::uint32_t const n_locations,
                                 gpu_vector_map<gpu_location_idx_t, gpu_u8_minutes> const* transfer_time,
                                 gpu_vecvec<gpu_location_idx_t, nigiri::gpu_footpath> const* gpu_footpaths_out,
                                 gpu_vecvec<gpu_location_idx_t, nigiri::gpu_footpath> const* gpu_footpaths_in_,
                                 gpu_raptor_stats* stats_){
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  for(auto l_idx = global_t_id;
       l_idx < n_locations; l_idx += global_stride){
    if(!marked(prev_station_mark_, l_idx)){
      continue;
    }
    auto const is_dest = is_dest_[l_idx];
    auto const tt = (dist_to_end_size_==0 && is_dest)
        ? 0 : dir<SearchDir>((*transfer_time)[gpu_location_idx_t{l_idx}]).count();
    const auto fp_target_time =
        static_cast<gpu_delta_t>(tmp_[l_idx] + tt);
    if(is_better<SearchDir>(fp_target_time, best_[l_idx])
        && is_better<SearchDir>(fp_target_time, time_at_dest_[k])){
      if(lb_[l_idx] == kUnreachable
          || !is_better<SearchDir>(fp_target_time + dir<SearchDir>(lb_[l_idx]), time_at_dest_[k])){
        ++stats_[l_idx%32].fp_update_prevented_by_lower_bound_;
        continue;
      }
      bool updated = update_arrival<SearchDir>(round_times_, k * column_count_round_times_ + l_idx, fp_target_time);
      if(updated){
        ++stats_[l_idx%32].n_earliest_arrival_updated_by_footpath_;
        best_[l_idx] = fp_target_time;
        mark(station_mark_, l_idx);
        if(is_dest){
          update_time_at_dest<SearchDir, Rt>(k, fp_target_time, time_at_dest_);
        }
      }
    }
  }
}

template <gpu_direction SearchDir, bool Rt>
__device__ void update_footpaths(unsigned const k, gpu_profile_idx_t const prf_idx, unsigned short kUnreachable,
                                 uint32_t* prev_station_mark_,
                                 gpu_delta_t* tmp_, gpu_delta_t* best_,
                                 uint16_t const* lb_, uint32_t* station_mark_, gpu_delta_t* time_at_dest_,
                                 bool const* is_dest_, gpu_delta_t* round_times_,
                                 uint32_t column_count_round_times_,
                                 std::uint32_t const n_locations,
                                 gpu_vector_map<gpu_location_idx_t, gpu_u8_minutes> const* transfer_time,
                                gpu_vecvec<gpu_location_idx_t, nigiri::gpu_footpath> const* gpu_footpaths_in,
                                 gpu_vecvec<gpu_location_idx_t, nigiri::gpu_footpath> const* gpu_footpaths_out,
                                 gpu_raptor_stats* stats_){
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  for(auto idx = global_t_id;
       idx < n_locations; idx += global_stride){
    if(!marked(prev_station_mark_, idx)){
      continue;
    }
    auto const l_idx = gpu_location_idx_t{idx};
    auto const& fps = (SearchDir == gpu_direction::kForward)
         ? gpu_footpaths_out[prf_idx][l_idx]
           : gpu_footpaths_in[prf_idx][l_idx];
    for(auto const& fp: fps){
      ++stats_[idx%32].n_footpaths_visited_;
      auto const target = gpu_to_idx(gpu_location_idx_t{fp.target_});
      auto const fp_target_time =
          gpu_clamp(tmp_[idx] + dir<SearchDir>(fp.duration()).count());

      if(is_better<SearchDir>(fp_target_time, best_[target])
          && is_better<SearchDir>(fp_target_time, time_at_dest_[k])){
        auto const lower_bound = lb_[gpu_to_idx(gpu_location_idx_t{fp.target_})];
        if(lower_bound == kUnreachable
            || !is_better<SearchDir>(fp_target_time + dir<SearchDir>(lower_bound), time_at_dest_[k])){
          ++stats_[idx%32].fp_update_prevented_by_lower_bound_;
          continue;
        }
      }
      bool updated = update_arrival<SearchDir>(round_times_, k * column_count_round_times_ +
                            gpu_to_idx(gpu_location_idx_t{fp.target_}), fp_target_time);
      if(updated){
        ++stats_[idx%32].n_earliest_arrival_updated_by_footpath_;
        best_[gpu_to_idx(gpu_location_idx_t{fp.target_})] = fp_target_time;
        mark(station_mark_, gpu_to_idx(gpu_location_idx_t{fp.target_}));
        if(is_dest_[gpu_to_idx(gpu_location_idx_t{fp.target_})]){
          update_time_at_dest<SearchDir, Rt>(k, fp_target_time, time_at_dest_);
        }
      }
    }
  }

}

template <gpu_direction SearchDir, bool Rt>
__device__ void update_intermodal_footpaths(unsigned const k, std::uint32_t const n_locations,
                                            uint16_t* dist_to_end_, uint32_t dist_to_end_size_, uint32_t* station_mark_,
                                            uint32_t* prev_station_mark_, gpu_delta_t* time_at_dest_,
                                            unsigned short kUnreachable, gpu_location_idx_t* gpu_kIntermodalTarget,
                                            gpu_delta_t* best_, gpu_delta_t* tmp_,
                                            gpu_delta_t* round_times_, uint32_t column_count_round_times_){
  if(dist_to_end_size_==0){
    return;
  }
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  if(global_t_id == 0){
    for(auto idx = 0U; idx != n_locations; ++idx){
      if((marked(prev_station_mark_, idx) || marked(station_mark_, idx)) && dist_to_end_[idx] != kUnreachable){
        auto const end_time = gpu_clamp(get_best<SearchDir>(best_[idx], tmp_[idx]) + dir<SearchDir>(dist_to_end_[idx]));
        if(is_better<SearchDir>(end_time, best_[gpu_to_idx(*gpu_kIntermodalTarget)])){
          bool updated = update_arrival<SearchDir>(round_times_, k * column_count_round_times_ +
                                           gpu_kIntermodalTarget->v_, end_time);
          best_[gpu_to_idx(*gpu_kIntermodalTarget)] = end_time;
          update_time_at_dest<SearchDir, Rt>(k, end_time, time_at_dest_);
        }
      }
    }
  }
}


template <gpu_direction SearchDir, bool Rt>
__device__ void raptor_round(unsigned const k, gpu_profile_idx_t const prf_idx,
                             gpu_day_idx_t* base_,
                             gpu_clasz_mask_t allowed_claszes_, uint16_t* dist_to_end_,
                             uint32_t dist_to_end_size_,
                             bool* is_dest_, uint16_t* lb_, int n_days_,
                             gpu_delta_t* time_at_dest_,
                             int* any_station_marked_, uint32_t* route_mark_,
                             uint32_t* station_mark_, gpu_delta_t* best_,
                             unsigned short kUnreachable, uint32_t* prev_station_mark_,
                             gpu_delta_t* round_times_, gpu_delta_t* tmp_,
                             uint32_t row_count_round_times_,
                             uint32_t column_count_round_times_,
                             gpu_location_idx_t* gpu_kIntermodalTarget,
                             gpu_raptor_stats* stats_, short* kMaxTravelTimeTicks_,
                             gpu_delta const* route_stop_times,
                             gpu_vecvec<gpu_route_idx_t,gpu_value_type> const* route_location_seq,
                             gpu_vecvec<gpu_location_idx_t , gpu_route_idx_t> const* location_routes,
                             std::uint32_t const n_locations,
                             std::uint32_t const n_routes,
                             gpu_vector_map<gpu_route_idx_t,gpu_interval<std::uint32_t>> const* route_stop_time_ranges,
                             gpu_vector_map<gpu_route_idx_t,gpu_interval<gpu_transport_idx_t >> const* route_transport_ranges,
                             gpu_vector_map<gpu_bitfield_idx_t, gpu_bitfield> const* bitfields,
                             gpu_vector_map<gpu_transport_idx_t,gpu_bitfield_idx_t> const* transport_traffic_days,
                             gpu_interval<gpu_sys_days> const* date_range,
                             gpu_vector_map<gpu_location_idx_t, gpu_u8_minutes> const* transfer_time,
                            gpu_vecvec<gpu_location_idx_t, nigiri::gpu_footpath> const* gpu_footpaths_in,
                             gpu_vecvec<gpu_location_idx_t, nigiri::gpu_footpath> const* gpu_footpaths_out,
                             gpu_vector_map<gpu_route_idx_t, gpu_clasz> const* route_clasz){
  // update_time_at_dest für alle locations
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  //TODO sicher, dass man über n_locations iterieren muss? -> aufpassen, dass round_times nicht out of range zugegriffen wird
  for(auto idx = global_t_id; idx < n_locations; idx += global_stride){
    auto test =round_times_[(k) * column_count_round_times_ +idx];
    auto test2 = best_[idx];
    best_[global_t_id] =get_best<SearchDir>(test, test2);
    if(is_dest_[idx]){
      update_time_at_dest<SearchDir, Rt>(k, best_[global_t_id], time_at_dest_);
    }
  }

  this_grid().sync();

  // für jede location & für jede location_route state_.route_mark_
  if(get_global_thread_id()==0){
    atomicAnd(any_station_marked_,0);
  }
  this_grid().sync();
  convert_station_to_route_marks<SearchDir, Rt>(station_mark_, route_mark_,
                                 any_station_marked_, location_routes, n_locations);

  this_grid().sync();

  if(!*any_station_marked_){
    if (get_global_thread_id() == 0) printf("GPU break k:%d",k);
    return;
  }

  swap_b_reset(prev_station_mark_,station_mark_,(n_locations/32)+1);

  this_grid().sync();
  if(get_global_thread_id() ==0) printf("GPU K: %d, any_marked %d",k,*any_station_marked_);
  (allowed_claszes_ == 0xffff)? loop_routes<SearchDir, Rt, false>(k, any_station_marked_, route_mark_, &allowed_claszes_,
                                                             stats_, kMaxTravelTimeTicks_, prev_station_mark_, best_,
                                                             round_times_, column_count_round_times_, tmp_, lb_, n_days_,
                                                             time_at_dest_, station_mark_, base_, kUnreachable,
                                                                 route_stop_times,
                                                                 route_location_seq,
                                                                 location_routes,
                                                                 n_locations,
                                                                 n_routes,
                                                                 route_stop_time_ranges,
                                                                 route_transport_ranges,
                                                                 bitfields,
                                                                 transport_traffic_days,
                                                                 date_range,
                                                                 transfer_time,
                                                                 gpu_footpaths_in,
                                                                 gpu_footpaths_out,
                                                                 route_clasz)
                           : loop_routes<SearchDir, Rt, true>(k, any_station_marked_, route_mark_, &allowed_claszes_,
                                                             stats_, kMaxTravelTimeTicks_, prev_station_mark_, best_,
                                                             round_times_, column_count_round_times_, tmp_, lb_, n_days_,
                                                             time_at_dest_, station_mark_, base_, kUnreachable,
                                                                route_stop_times,
                                                                route_location_seq,
                                                                location_routes,
                                                                n_locations,
                                                                n_routes,
                                                                route_stop_time_ranges,
                                                                route_transport_ranges,
                                                                bitfields,
                                                                transport_traffic_days,
                                                                date_range,
                                                                transfer_time,
                                                                gpu_footpaths_in,
                                                                gpu_footpaths_out,
                                                                route_clasz);
  this_grid().sync();

  if(!*any_station_marked_){
    if (get_global_thread_id() == 0) printf("GPU break2 k:%d",k);
    return;
  }

  reset_store(route_mark_,(n_routes/32)+1);

  swap_b_reset(prev_station_mark_,station_mark_,(n_locations/32)+1);

  this_grid().sync();

  // update_transfers
  update_transfers<SearchDir, Rt>(k, is_dest_, dist_to_end_, dist_to_end_size_,
                   tmp_, best_, time_at_dest_, kUnreachable, lb_, round_times_,
                                  column_count_round_times_, station_mark_, prev_station_mark_,
                   n_locations,
                                  transfer_time,
                                  gpu_footpaths_in,
                                  gpu_footpaths_out, stats_);
  this_grid().sync();



  // update_footpaths
  update_footpaths<SearchDir, Rt>(k, prf_idx, kUnreachable, prev_station_mark_,
                   tmp_, best_, lb_, station_mark_, time_at_dest_,
                   is_dest_, round_times_, column_count_round_times_,
                   n_locations,
                                  transfer_time,
                                  gpu_footpaths_in,
                                  gpu_footpaths_out, stats_);
  this_grid().sync();

  // update_intermodal_footpaths
  update_intermodal_footpaths<SearchDir, Rt>(k, n_locations, dist_to_end_, dist_to_end_size_, station_mark_,
                             prev_station_mark_, time_at_dest_, kUnreachable,
                             gpu_kIntermodalTarget, best_, tmp_, round_times_, column_count_round_times_);

}

template <gpu_direction SearchDir, bool Rt>
__device__ void init_arrivals(gpu_unixtime_t const worst_time_at_dest,
                              gpu_day_idx_t* base_, gpu_delta_t* time_at_dest,
                              gpu_delta const* route_stop_times,
                              gpu_vector_map<gpu_route_idx_t,gpu_interval<gpu_transport_idx_t >> const* route_transport_ranges,
                              gpu_interval<gpu_sys_days> const* date_range){
  auto const global_t_id = get_global_thread_id();
  auto const global_stride = get_global_stride();
  for(auto idx = global_t_id; idx <gpu_kMaxTransfers+1; idx += global_stride){
    time_at_dest[idx] = get_best<SearchDir>(unix_to_gpu_delta(base(base_,date_range), worst_time_at_dest), time_at_dest[idx]);
  }

}

// größten Teil von raptor.execute() wird hierdrin ausgeführt
// kernel muss sich außerhalb der gpu_raptor Klasse befinden
template <gpu_direction SearchDir, bool Rt>
__global__ void gpu_raptor_kernel(gpu_unixtime_t* start_time,
                                  uint8_t max_transfers,
                                  gpu_unixtime_t* worst_time_at_dest,
                                  gpu_profile_idx_t* prf_idx,
                                  gpu_clasz_mask_t* allowed_claszes,
                                  std::uint16_t* dist_to_end,
                                  std::uint32_t* dist_to_end_size,
                                  gpu_day_idx_t* base,
                                  bool* is_dest,
                                  std::uint16_t* lb,
                                  int* n_days,
                                  std::uint16_t* kUnreachable,
                                  gpu_location_idx_t* kIntermodalTarget,
                                  short* kMaxTravelTimeTicks,
                                  gpu_delta_t* tmp,
                                  gpu_delta_t* best,
                                  gpu_delta_t* round_times,
                                  gpu_delta_t* time_at_dest,
                                  uint32_t* station_mark,
                                  uint32_t* prev_station_mark,
                                  uint32_t* route_mark,
                                  int* any_station_marked,
                                  uint32_t row_count_round_times,
                                  uint32_t column_count_round_times,
                                  gpu_raptor_stats* stats,
                                  gpu_delta const* route_stop_times,
                                  gpu_vecvec<gpu_route_idx_t,gpu_value_type> const* route_location_seq,
                                  gpu_vecvec<gpu_location_idx_t , gpu_route_idx_t> const* location_routes,
                                  std::uint32_t const n_locations,
                                  std::uint32_t const n_routes,
                                  gpu_vector_map<gpu_route_idx_t,gpu_interval<std::uint32_t>> const* route_stop_time_ranges,
                                  gpu_vector_map<gpu_route_idx_t,gpu_interval<gpu_transport_idx_t >> const* route_transport_ranges,
                                  gpu_vector_map<gpu_bitfield_idx_t, gpu_bitfield> const* bitfields,
                                  gpu_vector_map<gpu_transport_idx_t,gpu_bitfield_idx_t> const* transport_traffic_days,
                                  gpu_interval<gpu_sys_days> const* date_range,
                                  gpu_vector_map<gpu_location_idx_t, gpu_u8_minutes> const* transfer_time,
                                  gpu_vecvec<gpu_location_idx_t, nigiri::gpu_footpath> const* gpu_footpaths_in,
                                  gpu_vecvec<gpu_location_idx_t, nigiri::gpu_footpath> const* gpu_footpaths_out,
                                  gpu_vector_map<gpu_route_idx_t, gpu_clasz> const* route_clasz){
  auto const end_k =
      get_smaller(max_transfers, gpu_kMaxTransfers) + 1U;
  // 1. Initialisierung
  init_arrivals<SearchDir, Rt>(*worst_time_at_dest, base,
                time_at_dest, route_stop_times,route_transport_ranges,date_range);

  this_grid().sync();
  //++stats[get_global_thread_id()>>5].n_routes_visited_; TODO: so ist out of range
  // ausprobieren, ob folgende daten noch weiter entschachtelt werden müssen
  //locations->gpu_footpaths_out_[1][1]; // hiervon sind auch gpu_footpaths_out und transfer_time betroffem
  // 2. Update Routes

  for (auto k = 1U; k != end_k; ++k) { // diese Schleife bleibt, da alle Threads in jede Runde gehen
    // Resultate aus lezter Runde von device in variable speichern?  //TODO: typen von kIntermodalTarget und dist_to_end_size falsch???
    if(k!= 1 && (!(*any_station_marked))){
      break;
    }
    raptor_round<SearchDir, Rt>(k, *prf_idx, base, *allowed_claszes,
                 dist_to_end, *dist_to_end_size, is_dest, lb, *n_days,
                 time_at_dest, any_station_marked, route_mark,
                 station_mark, best,
                 *kUnreachable, prev_station_mark,
                 round_times, tmp,
                                column_count_round_times,
                 column_count_round_times,
                 kIntermodalTarget, stats, kMaxTravelTimeTicks,route_stop_times,
                 route_location_seq,
                                location_routes,
                                n_locations,
                                n_routes,
                                route_stop_time_ranges,
                                route_transport_ranges,
                                bitfields,
                                transport_traffic_days,
                                date_range,
                                transfer_time,
                                gpu_footpaths_in,
                                gpu_footpaths_out,
                                route_clasz);
    this_grid().sync();
  }
  this_grid().sync();
}

#define XSTR(s) STR(s)
#define STR(s) #s

#define CUDA_CALL(call) \
    if ((code = (call)) != hipSuccess) {                     \
      printf("CUDA error: %s at " XSTR(call) " %s:%d\n",     \
             hipGetErrorString(code), __FILE__, __LINE__); \
      goto fail;                                            \
    }

#define CUDA_COPY_TO_DEVICE(type, target, source, size)                        \
    CUDA_CALL(hipMalloc(&(target), (size) * sizeof(type)))                          \
    CUDA_CALL(                                                                   \
        hipMemcpy(target, source, (size) * sizeof(type), hipMemcpyHostToDevice))

void copy_to_devices(gpu_clasz_mask_t const& allowed_claszes,
                     std::vector<std::uint16_t> const& dist_to_dest,
                     gpu_day_idx_t const& base,
                     std::vector<uint8_t> const& is_dest,
                     std::vector<std::uint16_t> const& lb,
                     int const& n_days,
                     std::uint16_t const& kUnreachable,
                     short const& kMaxTravelTimeTicks,
                     unsigned int const& kIntermodalTarget,
                     gpu_clasz_mask_t*& allowed_claszes_,
                     std::uint16_t* & dist_to_end_,
                     std::uint32_t* & dist_to_end_size_,
                     gpu_day_idx_t* & base_,
                     bool* & is_dest_,
                     std::uint16_t* & lb_,
                     int* & n_days_,
                     std::uint16_t* & kUnreachable_,
                     gpu_location_idx_t* & kIntermodalTarget_,
                     short* & kMaxTravelTimeTicks_){
  printf(" ");//DO NOT DELETE, SUPPRESSES Assertion failed: __acrt_first_block == header
  //Wahrscheinlich von übergeben Parametern das die nicht direkt richtig sind
  hipError_t code;
  auto dist_to_end_size = dist_to_dest.size();

  allowed_claszes_ = nullptr;
  CUDA_COPY_TO_DEVICE(gpu_clasz_mask_t, allowed_claszes_, &allowed_claszes, 1);
  dist_to_end_ = nullptr;
  CUDA_COPY_TO_DEVICE(std::uint16_t, dist_to_end_, dist_to_dest.data(),
                      dist_to_dest.size());
  dist_to_end_size_ = nullptr;
  CUDA_COPY_TO_DEVICE(std::uint32_t, dist_to_end_size_, &dist_to_end_size, 1);
  base_ = nullptr;
  CUDA_COPY_TO_DEVICE(gpu_day_idx_t, base_, &base, 1);
  is_dest_ = nullptr;
  CUDA_COPY_TO_DEVICE(bool, is_dest_, is_dest.data(), is_dest.size());
  lb_ = nullptr;
  CUDA_COPY_TO_DEVICE(std::uint16_t, lb_, lb.data(), lb.size());
  n_days_ = nullptr;
  CUDA_COPY_TO_DEVICE(int, n_days_, &n_days, 1);
  kUnreachable_ = nullptr;
  CUDA_COPY_TO_DEVICE(std::uint16_t, kUnreachable_, &kUnreachable, 1);
  kIntermodalTarget_ = nullptr;
  CUDA_COPY_TO_DEVICE(gpu_location_idx_t, kIntermodalTarget_,
                      &kIntermodalTarget, 1);
  kMaxTravelTimeTicks_ = nullptr;
  CUDA_COPY_TO_DEVICE(short, kMaxTravelTimeTicks_, &kMaxTravelTimeTicks, 1);
  return;
fail:
  hipFree(allowed_claszes_);
  hipFree(dist_to_end_);
  hipFree(dist_to_end_size_);
  hipFree(base_);
  hipFree(is_dest_);
  hipFree(lb_);
  hipFree(n_days_);
  hipFree(kUnreachable_);
  hipFree(kIntermodalTarget_);
  hipFree(kMaxTravelTimeTicks_);
  return;
};
void copy_to_device_destroy(
    gpu_clasz_mask_t*& allowed_claszes_,
    std::uint16_t* & dist_to_end_,
    std::uint32_t* & dist_to_end_size_,
    gpu_day_idx_t* & base_,
    bool* & is_dest_,
    std::uint16_t* & lb_,
    int* & n_days_,
    std::uint16_t* & kUnreachable_,
    gpu_location_idx_t* & kIntermodalTarget_,
    short* & kMaxTravelTimeTicks_){
  hipFree(allowed_claszes_);
  allowed_claszes_ = nullptr;
  hipFree(dist_to_end_);
  dist_to_end_ = nullptr;
  hipFree(dist_to_end_size_);
  dist_to_end_size_ = nullptr;
  hipFree(base_);
  base_ = nullptr;
  hipFree(is_dest_);
  is_dest_ = nullptr;
  hipFree(lb_);
  lb_ = nullptr;
  hipFree(n_days_);
  n_days_ = nullptr;
  hipFree(kUnreachable_);
  kUnreachable_ = nullptr;
  hipFree(kIntermodalTarget_);
  kIntermodalTarget_ = nullptr;
  hipFree(kMaxTravelTimeTicks_);
  kMaxTravelTimeTicks_ = nullptr;
  auto const last_error = hipGetLastError();
  if (last_error != hipSuccess) {
    printf("CUDA error: %s at " STR(last_error) " %s:%d\n",
           hipGetErrorString(last_error), __FILE__, __LINE__);
  }
};

void launch_kernel(void** args,
                          device_context const& device,
                          hipStream_t s,
                          gpu_direction search_dir,
                          bool rt) {
  hipSetDevice(device.id_);
  // Kernel-Auswahl basierend auf Parametern
  void* kernel_func = nullptr;
  if (search_dir == gpu_direction::kForward && rt == true) {
    kernel_func = (void*)gpu_raptor_kernel<gpu_direction::kForward, true>;
  } else if (search_dir == gpu_direction::kForward && rt == false) {
    kernel_func = (void*)gpu_raptor_kernel<gpu_direction::kForward, false>;
  } else if (search_dir == gpu_direction::kBackward && rt == true) {
    kernel_func = (void*)gpu_raptor_kernel<gpu_direction::kBackward, true>;
  } else if (search_dir == gpu_direction::kBackward && rt == false) {
    kernel_func = (void*)gpu_raptor_kernel<gpu_direction::kBackward, false>;
  }
  auto start_kernel = std::chrono::high_resolution_clock::now();
  hipLaunchCooperativeKernel(reinterpret_cast<const void*>(kernel_func), device.grid_, device.threads_per_block_, args, 0, s);
  hipDeviceSynchronize();
  auto end_kernel = std::chrono::high_resolution_clock::now();
  auto kernel_duration = std::chrono::duration_cast<std::chrono::microseconds>(end_kernel - start_kernel).count();
  std::cout << "Kernel Time: " << kernel_duration << " microseconds\n";
  cuda_check();
}

inline void fetch_arrivals_async(mem* mem, hipStream_t s) {
  hipMemcpyAsync(
      mem->host_.round_times_.data(), mem->device_.round_times_,
      sizeof(gpu_delta_t)*mem->host_.row_count_round_times_*mem->host_.column_count_round_times_, hipMemcpyDeviceToHost, s);
  cuda_check();
  hipMemcpyAsync(
      mem->host_.stats_.data(), mem->device_.stats_,
      sizeof(gpu_raptor_stats)*32, hipMemcpyDeviceToHost, s);
  cuda_check();
  hipMemcpyAsync(
      mem->host_.tmp_.data(), mem->device_.tmp_,
      sizeof(gpu_delta_t)*mem->device_.n_locations_, hipMemcpyDeviceToHost, s);
  cuda_check();
  hipMemcpyAsync(
      mem->host_.best_.data(), mem->device_.best_,
      sizeof(gpu_delta_t)*mem->device_.n_locations_, hipMemcpyDeviceToHost, s);
  cuda_check();
}
void copy_back(mem* mem){
  cuda_check();
  cuda_sync_stream(mem->context_.proc_stream_);
  fetch_arrivals_async(mem,mem->context_.transfer_stream_);
  cuda_check();
  cuda_sync_stream(mem->context_.transfer_stream_);
  cuda_check();
}

void add_start_gpu(std::vector<gpu_delta_t>& best, std::vector<gpu_delta_t>& round_times,std::vector<uint32_t>& station_mark,mem* mem){
  hipMemcpy(mem->device_.best_, best.data(), mem->device_.n_locations_ * sizeof(gpu_delta_t), hipMemcpyHostToDevice);
  hipMemcpy(mem->device_.round_times_, round_times.data(), round_times.size() * sizeof(gpu_delta_t), hipMemcpyHostToDevice);
  hipMemcpy(mem->device_.station_mark_, station_mark.data(), station_mark.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
}
std::unique_ptr<mem> gpu_mem(
    std::vector<gpu_delta_t>& tmp,
    std::vector<gpu_delta_t>& best,
    std::vector<bool>& station_mark,
    std::vector<bool>& prev_station_mark,
    std::vector<bool>& route_mark,
    gpu_direction search_dir,
    gpu_timetable const* gtt){

  short kInvalid = (search_dir == gpu_direction::kForward)
                       ? kInvalidGpuDelta<gpu_direction::kForward>
                       : kInvalidGpuDelta<gpu_direction::kBackward>;

  size_t num_uint32_locations = (gtt->n_locations_ / 32) + 1;

  std::vector<uint32_t> gpu_station_mark(num_uint32_locations, 0);
  std::vector<uint32_t> gpu_prev_station_mark(num_uint32_locations, 0);
  std::vector<uint32_t> gpu_route_mark((gtt->n_routes_ / 32) + 1, 0);

  size_t count = station_mark.size();
  for (size_t i = 0; i < count; ++i) {
    // Station Mark
    gpu_station_mark[i / 32] |= (station_mark[i] << (i % 32));

    // Previous Station Mark
    gpu_prev_station_mark[i / 32] |= (prev_station_mark[i] << (i % 32));
  }


  for (size_t i = 0; i < route_mark.size(); ++i) {
    if (route_mark[i]) {
      gpu_route_mark[i / 32] |= (1u << (i % 32));
    }
  }

  gpu_raptor_state state;
  state.init(*gtt, kInvalid);
  loaned_mem loan(state, kInvalid);
  std::unique_ptr<mem> mem = std::move(loan.mem_);

  hipMemcpy(mem->device_.tmp_, tmp.data(), gtt->n_locations_ * sizeof(gpu_delta_t), hipMemcpyHostToDevice);
  cuda_check();
  hipMemcpy(mem->device_.best_, best.data(), gtt->n_locations_ * sizeof(gpu_delta_t), hipMemcpyHostToDevice);
  cuda_check();
  hipMemcpy(mem->device_.station_mark_, gpu_station_mark.data(), num_uint32_locations * sizeof(uint32_t), hipMemcpyHostToDevice);
  cuda_check();
  hipMemcpy(mem->device_.prev_station_mark_, gpu_prev_station_mark.data(), num_uint32_locations * sizeof(uint32_t), hipMemcpyHostToDevice);
  cuda_check();
  hipMemcpy(mem->device_.route_mark_, gpu_route_mark.data(), ((gtt->n_routes_ / 32) + 1) * sizeof(uint32_t), hipMemcpyHostToDevice);
  cuda_check();

  hipDeviceSynchronize();
  return mem;
}

void copy_to_gpu_args(gpu_unixtime_t const* start_time,
                      gpu_unixtime_t const* worst_time_at_dest,
                      gpu_profile_idx_t const* prf_idx,
                      gpu_unixtime_t*& start_time_ptr,
                      gpu_unixtime_t*& worst_time_at_dest_ptr,
                      gpu_profile_idx_t*& prf_idx_ptr){
  hipError_t code;
  CUDA_COPY_TO_DEVICE(gpu_unixtime_t,start_time_ptr,start_time,1);
  CUDA_COPY_TO_DEVICE(gpu_unixtime_t,worst_time_at_dest_ptr,worst_time_at_dest,1);
  CUDA_COPY_TO_DEVICE(gpu_profile_idx_t ,prf_idx_ptr,prf_idx,1);
  return;
  fail:
    hipFree(start_time_ptr);
    hipFree(worst_time_at_dest_ptr);
    hipFree(prf_idx_ptr);
    return;
}
void destroy_copy_to_gpu_args(gpu_unixtime_t* start_time_ptr,
                              gpu_unixtime_t* worst_time_at_dest_ptr,
                              gpu_profile_idx_t* prf_idx_ptr){
  hipFree(start_time_ptr);
  start_time_ptr = nullptr;
  hipFree(worst_time_at_dest_ptr);
  worst_time_at_dest_ptr = nullptr;
  hipFree(prf_idx_ptr);
  prf_idx_ptr = nullptr;
  cuda_check();
}